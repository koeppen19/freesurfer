#include "hip/hip_runtime.h"
/**
 * @file  em_register_cuda.cu
 * @brief Holds em_register CUDA routines
 *
 * Contains CUDA routines for em_register
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/08/28 22:11:21 $
 *    $Revision: 1.5.2.2 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include <iostream>
#include <iomanip>
#include <memory>
#include <fstream>
#include <sstream>
#include <vector>
using namespace std;


#include <thrust/device_new_allocator.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>



#include "cudacheck.h"

#include "mriframegpu.hpp"
#include "affinegpu.hpp"
#include "gcasgpu.hpp"
#include "ctfactory.hpp"

#include "generators.hpp"
#include "cudatypeutils.hpp"

#include "em_register_cuda.h"

//#define OUTPUT_STAGES

#ifdef OUTPUT_STAGES
#include <netcdf.h>

#define NC_SAFE_CALL( call ) do {		\
    int err = call;				\
    if( NC_NOERR != err ) {			\
      std::cerr << __FUNCTION__ \
		<< ": NetCDF failure on line " << __LINE__	\
		<< " of file " << __FILE__			\
		<< std::endl;					\
      std::cerr << "Error code was " << err << std::endl;	\
      std::cerr << "Error string was : " << nc_strerror(err)	\
		<< std::endl;					\
      abort();                                                  \
    }								\
  } while ( 0 );




void WriteTranslationLogPs( const float minTrans,
                            const float maxTrans,
                            const unsigned int nTrans,
                            const float* const logps,
                            const unsigned int iteration ) {
  enum dimIndices{ iX, iY, iZ };

  // Create the filename
  std::stringstream fileName;
  fileName << "TranslationLogPs"
           << setw(4) << setfill( '0' ) << iteration
           << ".nc";

  // Set up the translations
  TranslationGenerator myGen( minTrans, maxTrans, nTrans );

  // Reference for the file
  int ncid;

  // Open the file
  NC_SAFE_CALL( nc_create( fileName.str().c_str(), NC_CLOBBER, &ncid ) );

  // Set up the dimensions
  int dimIDs[nTrans];
  NC_SAFE_CALL( nc_def_dim( ncid, "delta_x", nTrans, &dimIDs[iX] ) );
  NC_SAFE_CALL( nc_def_dim( ncid, "delta_y", nTrans, &dimIDs[iY] ) );
  NC_SAFE_CALL( nc_def_dim( ncid, "delta_z", nTrans, &dimIDs[iZ] ) );

  // Set up the variable ID
  int varID;
  NC_SAFE_CALL( nc_def_var( ncid,
                            "log_p",
                            NC_FLOAT,
                            3, dimIDs,
                            &varID ) );

  // Make the end of the 'definition' region
  NC_SAFE_CALL( nc_enddef( ncid ) );

  // Write the log ps
  NC_SAFE_CALL( nc_put_var_float( ncid,
				  varID,
				  logps ) );


  // Close the file
  NC_SAFE_CALL( nc_close( ncid ) );
}

#endif


// ==================================================================




static GPU::Classes::MRIframeGPU<unsigned char> src_uchar;
static std::auto_ptr<GPU::Classes::CTfactory> srcFactory;
texture<unsigned char, 3, hipReadModeElementType> dt_mri;  // 3D texture


static GPU::Classes::GCASampleGPU myGCAS;

const unsigned int  kCalcLogPKernelSize = 256;
const unsigned int kOptimiseBlockSize = 256;


const unsigned int nIndices = 9;
enum Indices{ iMinTrans=0, iMaxTrans=1, inTrans=2,
	      iMinScale=3, iMaxScale=4, inScale=5,
	      iMinRot=6, iMaxRot=7, inRot=8 };

__constant__ float dc_TransformParams[nIndices];





// =================================================================
// Device Utility functions


//! Extracts data from the MRI texture
__device__ float MRIlookup( const float3 r ) {
  /*!
    Performs a lookup into the MRI texture
  */

  // Offset by 0.5, since texture values are assumed to be at voxel centres

  return( tex3D( dt_mri, r.x+0.5f, r.y+0.5f, r.z+0.5f ) );
}



//! Computes the log_p value for a single point.
__device__ float ComputeLogP( const float val, const float mean,
			      const float prior, const float covar,
                              const float clamp ) {
  
  float det = covar;

  float v = val - mean;

  float log_p = - logf( sqrtf( det ) ) - 0.5f*( v*v / covar ) + logf( prior );

  // Assume that clamp is already -ve
  if( log_p < clamp ) {
    log_p = clamp;
  }

  return( log_p );
}



//! Routine to sum all the logps for a given transform in shared memory
__device__ float SumLogPs( const GPU::Classes::AffineTransShared &afTrans,
			   const GPU::Classes::GCASonGPU& gcas,
                           const float clamp ) {
  // The accumulator array for this block
  __shared__ float myLogps[kOptimiseBlockSize];
  myLogps[threadIdx.x] = 0;
  
  // Accumulate log p values in shared memory
  for( unsigned int i=0; i<gcas.nSamples; i+= kOptimiseBlockSize ) {
    if( (i + threadIdx.x) < gcas.nSamples ) {
      float3 rOut = afTrans.transform( gcas.GetLocation( i+threadIdx.x ) );

      float mriVal = MRIlookup( rOut );

      myLogps[threadIdx.x] += ComputeLogP( mriVal,
					   gcas.means[i+threadIdx.x],
					   gcas.priors[i+threadIdx.x],
					   gcas.covars[i+threadIdx.x],
                                           clamp );
    }
  }

  __syncthreads();

  // Perform reduction sum
#if 1
  // Slow but always correct version
  for( unsigned int d=blockDim.x / 2; d>0; d>>=1 ) {
    if( threadIdx.x < d ) { 
      myLogps[threadIdx.x] += myLogps[threadIdx.x+d];
    }
    __syncthreads();
  }
#else
  // Version optimised for a warpsize of 32
  for( unsigned int d=blockDim.x / 2; d>32; d>>=1 ) {
    if( threadIdx.x < d ) { 
      myLogps[threadIdx.x] += myLogps[threadIdx.x+d];
    }
    __syncthreads();
  }

  if( threadIdx.x < 32 ) {
    myLogps[threadIdx.x] += myLogps[threadIdx.x+32];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+16];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+8];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+4];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+2];
    myLogps[threadIdx.x] += myLogps[threadIdx.x+1];
  }
#endif

  return( myLogps[0] );
}




// ===================================================================
// Device Kernels


//! Kernel to compute log_p for a single transform array
__global__
void ComputeAllLogP( const GPU::Classes::AffineTransformation afTrans,
		     const GPU::Classes::GCASonGPU gcas,
                     const float clamp,
		     float *logps ) {
  /*!
    Driver kernel to compute the value of log_p for every sample.
    Uses the MRI texture and transform matrix stored in constant memory
  */

  const unsigned int iSample = (blockIdx.x*blockDim.x) + threadIdx.x;

  if( iSample >= gcas.nSamples ) {
    // Nothing to do
    return;
  }

  // Compute location with affine transform
  float3 rOut = afTrans.transform( gcas.GetLocation( iSample ) );

  float mriVal = MRIlookup( rOut );


  logps[iSample] = ComputeLogP( mriVal,
				gcas.means[iSample],
				gcas.priors[iSample],
				gcas.covars[iSample],
                                clamp );
}


//! Kernel to compute all probabilities for a given translation generator
__global__
void TranslationLogps( const GPU::Classes::AffineTransformation base,
                       const TranslationGenerator tGen,
                       const GPU::Classes::GCASonGPU gcas,
                       const float clamp,
                       float *logps ) {
  const size_t b1d = blockIdx.x + ( blockIdx.y * gridDim.x );

  // Find our translation
  float3 myTrans = tGen( b1d );
  
  __shared__ float m1[GPU::Classes::AffineTransShared::kMatrixSize];
  __shared__ float m2[GPU::Classes::AffineTransShared::kMatrixSize];

  GPU::Classes::AffineTransShared final( m1 ), translation( m2 );

  // Recast the input transformation (which will be in shared memory)
  // as an AffineTransShared. Unfortunately, this needs a const_cast
  const float* tmp = base.GetPointer();
  const GPU::Classes::AffineTransShared bAff( const_cast<float*>(tmp) );

  // Make identity transform
  translation.SetIdentity();

  // Note that we invert the translation
  translation.SetTranslation( -myTrans );

  // Reverse order of multiplications, as compared to mri_em_register.c
  final.Multiply( bAff, translation );
  
  __syncthreads();

  // -- All threads now have access to the transformation

  
  // Compute the final result

  float myLogP = SumLogPs( final, gcas, clamp );

  if( threadIdx.x == 0 ) {
    logps[ b1d ] = myLogP;
  }
}



//! Kernel to compute all transform probabilities for a given transform generator
__global__
void TransformLogps( const GPU::Classes::AffineTransformation base,
		     const float3 originTranslation,
		     const GPU::Classes::GCASonGPU gcas,
                     const float clamp,
		     float *logps ) {

  
  const long b1d = blockIdx.x + ( blockIdx.y * gridDim.x );

  // Find our transform
  LinearGenerator translate( dc_TransformParams[iMinTrans],
			     dc_TransformParams[iMaxTrans],
			     dc_TransformParams[inTrans] );
  LinearGenerator scale( dc_TransformParams[iMinScale],
			 dc_TransformParams[iMaxScale],
			 dc_TransformParams[inScale] );
  LinearGenerator rotate( dc_TransformParams[iMinRot],
			  dc_TransformParams[iMaxRot],
			  dc_TransformParams[inRot] );

  TransformGenerator tGen( translate, scale, rotate );

  float3 myTrans, myScale, myRot;

  tGen.GetTransform( b1d, myTrans, myScale, myRot );


  // Invert the transforms
  myTrans = -myTrans;
  myScale.x = 1/myScale.x;
  myScale.y = 1/myScale.y;
  myScale.z = 1/myScale.z;
  myRot = -myRot;


  // Compute the transform matrix
  __shared__ float m1[GPU::Classes::AffineTransShared::kMatrixSize];
  __shared__ float m2[GPU::Classes::AffineTransShared::kMatrixSize];
  __shared__ float m3[GPU::Classes::AffineTransShared::kMatrixSize];

  GPU::Classes::AffineTransShared A( m1 ), B( m2 ), C( m3 ) ;

  // Recast the input transformations (which will be in shared memory)
  // as an AffineTransShared. Unfortunately, this needs a const_cast
  const float* tmp = base.GetPointer();
  const GPU::Classes::AffineTransShared bAff( const_cast<float*>(tmp) );
  
  // Invert the order in the original find_optimal_linear_xform routine
  B.SetIdentity();
  B.SetTranslation( originTranslation );

  A.Multiply( bAff, B );
  
  B.SetIdentity();
  B.SetXRotation( myRot.x );
  
  C.Multiply( A, B );

  A.SetIdentity();
  A.SetYRotation( myRot.y );

  B.Multiply( C, A );

  A.SetIdentity();
  A.SetZRotation( myRot.z );
  
  C.Multiply( B, A );

  A.SetIdentity();
  A.SetScaling( myScale );

  B.Multiply( C, A );

  A.SetIdentity();
  A.SetTranslation( -originTranslation );

  C.Multiply( B, A );

  A.SetIdentity();
  A.SetTranslation( myTrans );

  B.Multiply( C, A );

  __syncthreads();

  // -- All threads now have access to the transformation
  
  float myLogp = SumLogPs( B, gcas, clamp );

  // Write the final result
  if( threadIdx.x == 0 ) {
    logps[ b1d ] = myLogp;
  }
}





// ===================================================================
// External Functions

float CUDA_ComputeLogSampleProbability( const MATRIX *m_L,
                                        const float clamp ) {
/*!
    Re-implementation of local_GCAcomputeLogSampleProbability() from
    file mri_em_register.c.
    Assumes that robust is set to false, making the original routine
    a pass-through to GCAcomputeLogSampleProbability() from file
    gca.c.
    Also assumed that CUDA_em_register_Prepare has already
    set up everything
  */

  thrust::device_ptr<float> d_logpvals;
  MATRIX *inv_m_L = NULL;

  // Get the inverse of the transform matrix
  inv_m_L = MatrixInverse( (MATRIX*)m_L, inv_m_L );

  GPU::Classes::AffineTransformation myTransform( inv_m_L );
  const GPU::Classes::GCASonGPU myGCASonGPU( myGCAS );
  const unsigned int nsamples = myGCASonGPU.nSamples;

  d_logpvals = thrust::device_new<float>( nsamples );

  // ---------------------------------------
  float logps;

  // Run the log_p evaluation kernel
  dim3 grid, threads;
  threads.x = kCalcLogPKernelSize;
  threads.y = threads.z = 1;
  grid.x = static_cast<int>( ceilf ( static_cast<float>(nsamples) / threads.x ) );
  grid.y = grid.z = 1;

  // Do the computation (note sign change on clamp!
  ComputeAllLogP<<<grid,threads>>>( myTransform, myGCASonGPU,
                                    -clamp,
				    thrust::raw_pointer_cast(d_logpvals) );
  CUDA_CHECK_ERROR( "ComputeAllLogP kernel failed!\n" );

  // Do the reduction
  logps = thrust::reduce( d_logpvals, d_logpvals+nsamples );

  MatrixFree( &inv_m_L );
  thrust::device_delete( d_logpvals );

  return( logps );
}





void CUDA_FindOptimalTranslation( const MATRIX *baseTransform,
				  const float minTrans,
				  const float maxTrans,
				  const unsigned int nTrans,
                                  const float clamp,
				  float *maxLogP,
				  float *dx,
				  float *dy,
				  float *dz ) {
  /*!
    Routine to find the best translation to match the
    MRI to the classifier array.
    A 'base' transform is supplied, and then translations
    within the given limits are searched
  */

#ifdef OUTPUT_STAGES
  static unsigned int nCalls = 0;

  std::stringstream fileName;
  fileName << "FastTransGPU"
           << std::setw(2) << std::setfill('0')
           << nCalls
           << ".output";
  
  std::ofstream outFile( fileName.str().c_str() );
#endif

  const unsigned int totalTrans = nTrans * nTrans * nTrans;

  // Device vector to hold logps
  thrust::device_ptr<float> d_logps;

  d_logps = thrust::device_new<float>( totalTrans );

  // Construct the generator which will give the required translations
  TranslationGenerator myGen( minTrans, maxTrans, nTrans );
  
  

  // Extract the 'base' transform, inverting
  MATRIX *invBaseTransform = NULL;
  invBaseTransform = MatrixInverse( baseTransform, invBaseTransform );
  GPU::Classes::AffineTransformation myBaseTransform( invBaseTransform );

  // Get the GCAsample array, which must already be on the GPU
  const GPU::Classes::GCASonGPU myGCASonGPU( myGCAS );

  // Compute all the probabilities
  dim3 grid, threads;

  threads.x = kOptimiseBlockSize;
  threads.y = threads.z = 1;

  grid.x = nTrans*nTrans;
  grid.y = nTrans;
  grid.z = 1;

  TranslationLogps<<<grid,threads>>>( myBaseTransform,
				      myGen,
				      myGCAS,
                                      -clamp, // Note sign change
				      thrust::raw_pointer_cast( d_logps ) );
  CUDA_CHECK_ERROR( "TranslationLogps failed!" );
 
#ifdef OUTPUT_STAGES
  std::vector<float> h_logps( totalTrans );
  for( unsigned int i=0; i<totalTrans; i++ ) {
    float3 translation = myGen(i);
    outFile << setw(20) << setprecision(12) << translation.x << ",";
    outFile << setw(20) << setprecision(12) << translation.y << ",";
    outFile << setw(20) << setprecision(12) << translation.z << ",";
    outFile << setw(20) << setprecision(12) << d_logps[i];
    outFile << "\n";
    h_logps.at(i) = d_logps[i];
  }
  WriteTranslationLogPs( minTrans, maxTrans, nTrans,
                         &h_logps[0], nCalls );
                         
#endif

  // Extract the maximum location
  thrust::device_ptr<float> maxLoc;
  maxLoc = thrust::max_element( d_logps, d_logps+totalTrans );

  // Get the maximum value
  *maxLogP = *maxLoc;

  // Convert the location to the required translation
  const int index = (maxLoc - d_logps);
  const float3 trans = myGen(index);


  *dx = trans.x;
  *dy = trans.y;
  *dz = trans.z;

  thrust::device_delete( d_logps );

#ifdef OUTPUT_STAGES
  nCalls++;
#endif

}





void CUDA_FindOptimalTransform( const MATRIX *baseTransform,
				const MATRIX *originTranslation,
				const float minTrans,
				const float maxTrans,
				const unsigned int nTrans,
				const float minScale,
				const float maxScale,
				const unsigned nScale,
				const float minRot,
				const float maxRot,
				const unsigned int nRot,
                                const float clamp,
				double *maxLogP,
				double *dx,
				double *dy,
				double *dz,
				double *sx,
				double *sy,
				double *sz,
				double *rx,
				double *ry,
				double *rz ) {
  /*!
    Routine to find the best transformation to match the
    MRI to the classifier array.
    A 'base' transform is supplied, and then transforms
    within the given limits are searched
  */

  const unsigned int totTranslate = nTrans * nTrans * nTrans;
  const unsigned int totOther = nScale * nScale * nScale *
    nRot * nRot * nRot;

  const unsigned int totalTransforms = totTranslate * totOther;

  // Device vector to hold logps
  thrust::device_ptr<float> d_logps;

  d_logps = thrust::device_new<float>( totalTransforms );

  // Construct the generator which will give the required translations
  LinearGenerator translate( minTrans, maxTrans, nTrans );
  LinearGenerator scale( minScale, maxScale, nScale );
  LinearGenerator rotate( minRot, maxRot, nRot );
  TransformGenerator myGen( translate, scale, rotate );

  // Get the transform generation parameters to the device
  float tParams[nIndices];

  tParams[iMinTrans] = minTrans;
  tParams[iMaxTrans] = maxTrans;
  tParams[inTrans] = nTrans;
  tParams[iMinScale] = minScale;
  tParams[iMaxScale] = maxScale;
  tParams[inScale] = nScale;
  tParams[iMinRot] = minRot;
  tParams[iMaxRot] = maxRot;
  tParams[inRot] = nRot;

  CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "dc_TransformParams"),
				      tParams,
				      nIndices*sizeof(float),
				      0,
				      hipMemcpyHostToDevice ) );

  // Extract the 'base' transform, inverting
  MATRIX *invBaseTransform = NULL;
  invBaseTransform = MatrixInverse( baseTransform, invBaseTransform );
  GPU::Classes::AffineTransformation myBaseTransform( invBaseTransform );

  // Extract the origin translation
  const float3 oTranslate = make_float3( originTranslation->rptr[1][4],
					 originTranslation->rptr[2][4],
					 originTranslation->rptr[3][4] );
  
  // Get the GCAsample array, which must already be on the GPU
  const GPU::Classes::GCASonGPU myGCASonGPU( myGCAS );

  dim3 grid, threads;

  // Compute all the probabilities
  threads.x = kOptimiseBlockSize;
  threads.y = threads.z = 1;

  grid.x = totTranslate;
  grid.y = totOther;
  grid.z = 1;

  TransformLogps<<<grid,threads>>>( myBaseTransform,
				    oTranslate,
				    myGCAS,
                                    -clamp, // Note sign change
				    thrust::raw_pointer_cast( d_logps ) );
  CUDA_CHECK_ERROR( "TransformLogps failed!" );
  
  // Locate the maximum probability
  thrust::device_ptr<float> maxLoc;
  maxLoc = thrust::max_element( d_logps, d_logps+totalTransforms );

  // Extract the maximum probability
  *maxLogP = *maxLoc;


  // Convert the maximum probability to the transform parameters
  const int index = (maxLoc - d_logps);

  float3 myTrans, myScale, myRot;
  myGen.GetTransform( index, myTrans, myScale, myRot );

  *dx = myTrans.x;
  *dy = myTrans.y;
  *dz = myTrans.z;

  *sx = myScale.x;
  *sy = myScale.y;
  *sz = myScale.z;

  *rx = myRot.x;
  *ry = myRot.y;
  *rz = myRot.z;

  thrust::device_delete( d_logps );

}




// =================================================================


void CUDA_em_register_Prepare( GCA *gca,
			       GCA_SAMPLE *gcas,
			       const MRI *mri,
			       const int nSamples ) {


  // Sanity check
  if( gca->ninputs != 1 ) {
    cerr << __FUNCTION__ << ": Must have ninputs==1" << endl;
    exit( EXIT_FAILURE );
  }

  // ------------------------------
  // Send the MRI
  const unsigned int nFrame = 0;
  
  src_uchar.Allocate( mri );
  src_uchar.Send( mri, nFrame );

  srcFactory.reset( new GPU::Classes::CTfactory( src_uchar,
                                                 dt_mri,
                                                 hipFilterModePoint ) );

  // Send the GCAS

  myGCAS.SendGPU( gca, gcas, mri, nSamples );

}


void CUDA_em_register_Release( void ) {
  CUDA_SAFE_CALL( hipUnbindTexture( dt_mri ) );
}
