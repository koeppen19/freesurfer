/**
 * @file  mriframegpu.cu
 * @brief Holds MRI frame template for the GPU
 *
 * Holds template specialisations for MRI frame on the GPU
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:08 $
 *    $Revision: 1.7.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */


#include "mriframegpu.hpp"

namespace GPU
{

namespace Classes
{

// ====================================================

template<> int MRIframeGPU<unsigned char>::MRItype( void ) const
{
  return( MRI_UCHAR );
}

template<> int MRIframeGPU<short>::MRItype( void ) const
{
  return( MRI_SHORT );
}

template<> int MRIframeGPU<float>::MRItype( void ) const
{
  return( MRI_FLOAT );
}

template<> int MRIframeGPU<int>::MRItype( void ) const
{
  return( MRI_INT );
}


// ====================================================

template<>
void MRIframeGPU<unsigned char>::ExhumeRow( const MRI* src,
    unsigned char* h_slab,
    const unsigned int iy,
    const unsigned int iz,
    const unsigned int iFrame ) const
{
  // Do the copy
  memcpy( h_slab,
          &MRIseq_vox( src, 0, iy, iz, iFrame ),
          src->width*sizeof(unsigned char) );
}

// -----

template<>
void MRIframeGPU<short>::ExhumeRow( const MRI* src,
                                    short* h_slab,
                                    const unsigned int iy,
                                    const unsigned int iz,
                                    const unsigned int iFrame ) const
{
  // Do the copy
  memcpy( h_slab,
          &MRISseq_vox( src, 0, iy, iz, iFrame ),
          src->width*sizeof(short) );
}

// -----

template<>
void MRIframeGPU<float>::ExhumeRow( const MRI* src,
                                    float* h_slab,
                                    const unsigned int iy,
                                    const unsigned int iz,
                                    const unsigned int iFrame ) const
{

  // Do the copy
  memcpy( h_slab,
          &MRIFseq_vox( src, 0, iy, iz, iFrame ),
          src->width*sizeof(float) );
}

// -----

template<>
void MRIframeGPU<int>::ExhumeRow( const MRI* src,
                                  int* h_slab,
                                  const unsigned int iy,
                                  const unsigned int iz,
                                  const unsigned int iFrame ) const
{

  // Do the copy
  memcpy( h_slab,
          &MRIIseq_vox( src, 0, iy, iz, iFrame ),
          src->width*sizeof(int) );
}



// ====================================================


template<>
void MRIframeGPU<unsigned char>::InhumeRow( MRI* dst,
    const unsigned char* h_slab,
    const unsigned int iy,
    const unsigned int iz,
    const unsigned int iFrame ) const
{
  // Do the copy
  memcpy( &MRIseq_vox( dst, 0, iy, iz, iFrame ),
          h_slab,
          dst->width*sizeof(unsigned char) );
}

// -----

template<>
void MRIframeGPU<short>::InhumeRow( MRI* dst,
                                    const short* h_slab,
                                    const unsigned int iy,
                                    const unsigned int iz,
                                    const unsigned int iFrame ) const
{
  // Do the copy
  memcpy( &MRISseq_vox( dst, 0, iy, iz, iFrame ),
          h_slab,
          dst->width*sizeof(short) );
}

// -----

template<>
void MRIframeGPU<float>::InhumeRow( MRI* dst,
                                    const float* h_slab,
                                    const unsigned int iy,
                                    const unsigned int iz,
                                    const unsigned int iFrame ) const
{
  // Do the copy
  memcpy( &MRIFseq_vox( dst, 0, iy, iz, iFrame ),
          h_slab,
          dst->width*sizeof(float) );
}

// -----

template<>
void MRIframeGPU<int>::InhumeRow( MRI* dst,
                                  const int* h_slab,
                                  const unsigned int iy,
                                  const unsigned int iz,
                                  const unsigned int iFrame ) const
{
  // Do the copy
  memcpy( &MRIIseq_vox( dst, 0, iy, iz, iFrame ),
          h_slab,
          dst->width*sizeof(int) );
}


}
}
