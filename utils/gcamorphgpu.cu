
#include <hip/hip_runtime.h>
/**
 * @file  gcamorphgpu.cu
 * @brief Holds GCA morph data on the GPU
 *
 *
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:07 $
 *    $Revision: 1.54.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */

#ifdef GCAMORPH_ON_GPU

#include "macros.h"

#include "chronometer.hpp"

#include "volumegpucompare.hpp"

#include "mriframegpu.hpp"
#include "mriconvolve_cuda.hpp"

#include "ctfactory.hpp"

#include "gcamorphgpu.hpp"


//! Texture reference for rx
texture<float,3,cudaReadModeElementType> dt_rx;
//! Texture reference for ry
texture<float,3,cudaReadModeElementType> dt_ry;
//! Texture reference for rz
texture<float,3,cudaReadModeElementType> dt_rz;

//! Texture reference for dx
texture<float,3,cudaReadModeElementType> dt_dx;
//! Texture reference for dy
texture<float,3,cudaReadModeElementType> dt_dy;
//! Texture reference for dz
texture<float,3,cudaReadModeElementType> dt_dz;


// ==============================================================

namespace GPU
{
namespace Classes
{

// --------------------------------------------

void GCAmorphGPU::CheckIntegrity( void ) const
{
  /*!
  Checks that all the allocated member arrays have
  the same dimensions.
  Aborts the program if the check fails
  */

  const dim3 myDims = this->d_rx.GetDims();

  bool good = ( myDims == this->d_ry.GetDims() );
  good = ( good && ( myDims == this->d_rz.GetDims() ) );

  good = ( good && ( myDims == this->d_origx.GetDims() ) );
  good = ( good && ( myDims == this->d_origy.GetDims() ) );
  good = ( good && ( myDims == this->d_origz.GetDims() ) );

  good = ( good && ( myDims == this->d_dx.GetDims() ) );
  good = ( good && ( myDims == this->d_dy.GetDims() ) );
  good = ( good && ( myDims == this->d_dz.GetDims() ) );

  good = ( good && ( myDims == this->d_odx.GetDims() ) );
  good = ( good && ( myDims == this->d_ody.GetDims() ) );
  good = ( good && ( myDims == this->d_odz.GetDims() ) );

  good = ( good && ( myDims == this->d_invalid.GetDims() ) );

  good = ( good && ( myDims == this->d_origArea.GetDims() ) );
  good = ( good && ( myDims == this->d_origArea1.GetDims() ) );
  good = ( good && ( myDims == this->d_origArea2.GetDims() ) );

  good = ( good && ( myDims == this->d_area.GetDims() ) );
  good = ( good && ( myDims == this->d_area1.GetDims() ) );
  good = ( good && ( myDims == this->d_area2.GetDims() ) );

  good = ( good && ( myDims == this->d_label.GetDims() ) );
  good = ( good && ( myDims == this->d_status.GetDims() ) );

  good = ( good && ( myDims == this->d_mean.GetDims() ) );
  good = ( good && ( myDims == this->d_variance.GetDims() ) );

  good = ( good && ( myDims == this->d_labelDist.GetDims() ) );

  // Work on the non-saved volumes
  good = ( good && ( myDims == this->d_xs.GetDims() ) );
  good = ( good && ( myDims == this->d_ys.GetDims() ) );
  good = ( good && ( myDims == this->d_zs.GetDims() ) );
  good = ( good && ( myDims == this->d_xs2.GetDims() ) );
  good = ( good && ( myDims == this->d_ys2.GetDims() ) );
  good = ( good && ( myDims == this->d_zs2.GetDims() ) );
  good = ( good && ( myDims == this->d_saved_origx.GetDims() ) );
  good = ( good && ( myDims == this->d_saved_origy.GetDims() ) );
  good = ( good && ( myDims == this->d_saved_origz.GetDims() ) );

  if( !good )
  {
    std::cerr << __FUNCTION__
              << ": Dimension mismatch"
              << std::endl;
    abort();
  }
}

// --------------------------------------------

void GCAmorphGPU::AllocateAll( const dim3& dims )
{
  /*!
  Allocates GPU memory to hold a volume
  of the given size.
  If possible, it keeps the current allocation.
  */

  // Start by seeing if the current allocation is consistent
  this->CheckIntegrity();

  // See if we can re-use existing allocation
  if( dims == this->d_rx.GetDims() )
  {
    return;
  }

  // Release existing memory
  this->ReleaseAll();

  // Allocate anew
  this->d_rx.Allocate( dims );
  this->d_ry.Allocate( dims );
  this->d_rz.Allocate( dims );

  this->d_origx.Allocate( dims );
  this->d_origy.Allocate( dims );
  this->d_origz.Allocate( dims );

  this->d_dx.Allocate( dims );
  this->d_dy.Allocate( dims );
  this->d_dz.Allocate( dims );

  this->d_odx.Allocate( dims );
  this->d_ody.Allocate( dims );
  this->d_odz.Allocate( dims );

  this->d_area.Allocate( dims );
  this->d_area1.Allocate( dims );
  this->d_area2.Allocate( dims );

  this->d_origArea.Allocate( dims );
  this->d_origArea1.Allocate( dims );
  this->d_origArea2.Allocate( dims );

  this->d_invalid.Allocate( dims );
  this->d_label.Allocate( dims );
  this->d_status.Allocate( dims );
  this->d_labelDist.Allocate( dims );

  this->d_mean.Allocate( dims );
  this->d_variance.Allocate( dims );

  // The non-saved volumes
  this->d_xs.Allocate( dims );
  this->d_ys.Allocate( dims );
  this->d_zs.Allocate( dims );
  this->d_xs2.Allocate( dims );
  this->d_ys2.Allocate( dims );
  this->d_zs2.Allocate( dims );
  this->d_saved_origx.Allocate( dims );
  this->d_saved_origy.Allocate( dims );
  this->d_saved_origz.Allocate( dims );
}


void GCAmorphGPU::ReleaseAll( void )
{
  /*!
  Releases each of the members.
  Recall that the VolumeGPU::Release method
  will also release any CUDA arrays.
  */
  this->d_rx.Release();
  this->d_ry.Release();
  this->d_rz.Release();

  this->d_dx.Release();
  this->d_dy.Release();
  this->d_dz.Release();

  this->d_odx.Release();
  this->d_ody.Release();
  this->d_odz.Release();

  this->d_origx.Release();
  this->d_origy.Release();
  this->d_origz.Release();

  this->d_origArea.Release();
  this->d_origArea1.Release();
  this->d_origArea2.Release();

  this->d_area.Release();
  this->d_area1.Release();
  this->d_area2.Release();

  this->d_invalid.Release();
  this->d_label.Release();
  this->d_status.Release();
  this->d_labelDist.Release();

  this->d_mean.Release();
  this->d_variance.Release();

  // The non-saved volumes
  this->d_xs.Release();
  this->d_ys.Release();
  this->d_zs.Release();
  this->d_xs2.Release();
  this->d_ys2.Release();
  this->d_zs2.Release();
  this->d_saved_origx.Release();
  this->d_saved_origy.Release();
  this->d_saved_origz.Release();

}



void GCAmorphGPU::ClearAll( void )
{
  this->d_rx.Zero();
  this->d_ry.Zero();
  this->d_rz.Zero();

  this->d_dx.Zero();
  this->d_dy.Zero();
  this->d_dz.Zero();

  this->d_odx.Zero();
  this->d_ody.Zero();
  this->d_odz.Zero();

  this->d_origx.Zero();
  this->d_origy.Zero();
  this->d_origz.Zero();

  this->d_origArea.Zero();
  this->d_origArea1.Zero();
  this->d_origArea2.Zero();

  this->d_area.Zero();
  this->d_area1.Zero();
  this->d_area2.Zero();

  this->d_invalid.Zero();
  this->d_label.Zero();
  this->d_status.Zero();
  this->d_labelDist.Zero();

  this->d_mean.Zero();
  this->d_variance.Zero();

  // The non-saved volumes
  this->d_xs.Zero();
  this->d_ys.Zero();
  this->d_zs.Zero();
  this->d_xs2.Zero();
  this->d_ys2.Zero();
  this->d_zs2.Zero();
  this->d_saved_origx.Zero();
  this->d_saved_origy.Zero();
  this->d_saved_origz.Zero();
}


// --------------------------------------------

void GCAmorphGPU::SendAll( const GCAM* src )
{
  /*!
  Sends all supported data in the given GCAM
  to the GPU.
  This involves a lot of packing data, and hence
  is going to be painfully slow
  */

  GCAmorphGPU::tSendTot.Start();

  this->CheckIntegrity();

#if 0
  std::cerr << __FUNCTION__
            << ": Catching gcamorph usage"
            << std::endl;
  exit( EXIT_FAILURE );
#endif

  // Check for number of inputs
  if( src->ninputs != 1 )
  {
    std::cerr << __FUNCTION__
              << ": Must have only one input in the GC1D!"
              << std::endl;
    exit( EXIT_FAILURE );
  }

  // Copy scalars
  this->exp_k = src->exp_k;
  this->neg = src->neg;
  this->gca = src->gca;
  this->spacing = src->spacing;

  // Extract the dimensions
  const dim3 dims = make_uint3( src->width,
                                src->height,
                                src->depth );

  // Allocate device memory
  this->AllocateAll( dims );

  // Allocate some page-locked host buffers
  GCAmorphGPU::AllocateHost( *this );


  GCAmorphGPU::tSendPack.Start();
  for( unsigned int i=0; i<dims.x; i++ )
  {
    for( unsigned int j=0; j<dims.y; j++ )
    {
      for( unsigned int k=0; k<dims.z; k++ )
      {

        // Get the 1d index (same for all arrays)
        const unsigned int i1d = this->d_rx.Index1D( i, j, k );
        // Get the current node
        const GCA_MORPH_NODE& gcamn = src->nodes[i][j][k];

        // Pack the data
        GCAmorphGPU::h_rx[i1d] = gcamn.x;
        GCAmorphGPU::h_ry[i1d] = gcamn.y;
        GCAmorphGPU::h_rz[i1d] = gcamn.z;

        GCAmorphGPU::h_origx[i1d] = gcamn.origx;
        GCAmorphGPU::h_origy[i1d] = gcamn.origy;
        GCAmorphGPU::h_origz[i1d] = gcamn.origz;

        GCAmorphGPU::h_dx[i1d] = gcamn.dx;
        GCAmorphGPU::h_dy[i1d] = gcamn.dy;
        GCAmorphGPU::h_dz[i1d] = gcamn.dz;

        GCAmorphGPU::h_odx[i1d] = gcamn.odx;
        GCAmorphGPU::h_ody[i1d] = gcamn.ody;
        GCAmorphGPU::h_odz[i1d] = gcamn.odz;

        GCAmorphGPU::h_origArea[i1d] = gcamn.orig_area;
        GCAmorphGPU::h_origArea1[i1d] = gcamn.orig_area1;
        GCAmorphGPU::h_origArea2[i1d] = gcamn.orig_area2;

        GCAmorphGPU::h_area[i1d] = gcamn.area;
        GCAmorphGPU::h_area1[i1d] = gcamn.area1;
        GCAmorphGPU::h_area2[i1d] = gcamn.area2;

        GCAmorphGPU::h_invalid[i1d] = gcamn.invalid;
        GCAmorphGPU::h_status[i1d] = gcamn.status;
        GCAmorphGPU::h_label[i1d] = gcamn.label;
        GCAmorphGPU::h_labelDist[i1d] = gcamn.label_dist;

        // Deal with the GC1D
        if( gcamn.gc != NULL )
        {
          /*
          Store the mean and variance.
          Check at top of the routine has ensured
          that there's only one input.
          This means that the covariance is really
          a variance
          */
          GCAmorphGPU::h_mean[i1d] = gcamn.gc->means[0];
          GCAmorphGPU::h_variance[i1d] = gcamn.gc->covars[0];
        }
        else
        {
          /*
          Store negative numbers to indicate that
          there is no GC1D here.
          Since a variance must be >=0, this is
          a reliable test
          */
          GCAmorphGPU::h_mean[i1d] = -1;
          GCAmorphGPU::h_variance[i1d] = -1;
        }

        // Deal with the non-saved members
        GCAmorphGPU::h_xs[i1d] = gcamn.xs;
        GCAmorphGPU::h_ys[i1d] = gcamn.ys;
        GCAmorphGPU::h_zs[i1d] = gcamn.zs;
        GCAmorphGPU::h_xs2[i1d] = gcamn.xs2;
        GCAmorphGPU::h_ys2[i1d] = gcamn.ys2;
        GCAmorphGPU::h_zs2[i1d] = gcamn.zs2;
        GCAmorphGPU::h_saved_origx[i1d] = gcamn.saved_origx;
        GCAmorphGPU::h_saved_origy[i1d] = gcamn.saved_origy;
        GCAmorphGPU::h_saved_origz[i1d] = gcamn.saved_origz;



      }
    }
  }
  GCAmorphGPU::tSendPack.Stop();


  GCAmorphGPU::tSendTransfer.Start();
  // Send the data
  this->d_rx.SendBuffer( GCAmorphGPU::h_rx );
  this->d_ry.SendBuffer( GCAmorphGPU::h_ry );
  this->d_rz.SendBuffer( GCAmorphGPU::h_rz );

  this->d_origx.SendBuffer( GCAmorphGPU::h_origx );
  this->d_origy.SendBuffer( GCAmorphGPU::h_origy );
  this->d_origz.SendBuffer( GCAmorphGPU::h_origz );

  this->d_dx.SendBuffer( GCAmorphGPU::h_dx );
  this->d_dy.SendBuffer( GCAmorphGPU::h_dy );
  this->d_dz.SendBuffer( GCAmorphGPU::h_dz );

  this->d_odx.SendBuffer( GCAmorphGPU::h_odx );
  this->d_ody.SendBuffer( GCAmorphGPU::h_ody );
  this->d_odz.SendBuffer( GCAmorphGPU::h_odz );

  this->d_origArea.SendBuffer( GCAmorphGPU::h_origArea );
  this->d_origArea1.SendBuffer( GCAmorphGPU::h_origArea1 );
  this->d_origArea2.SendBuffer( GCAmorphGPU::h_origArea2 );

  this->d_area.SendBuffer( GCAmorphGPU::h_area );
  this->d_area1.SendBuffer( GCAmorphGPU::h_area1 );
  this->d_area2.SendBuffer( GCAmorphGPU::h_area2 );

  this->d_invalid.SendBuffer( GCAmorphGPU::h_invalid );
  this->d_status.SendBuffer( GCAmorphGPU::h_status );
  this->d_label.SendBuffer( GCAmorphGPU::h_label );
  this->d_labelDist.SendBuffer( GCAmorphGPU::h_labelDist );

  this->d_mean.SendBuffer( GCAmorphGPU::h_mean );
  this->d_variance.SendBuffer( GCAmorphGPU::h_variance );

  // And the non-saved variables
  this->d_xs.SendBuffer( GCAmorphGPU::h_xs );
  this->d_ys.SendBuffer( GCAmorphGPU::h_ys );
  this->d_zs.SendBuffer( GCAmorphGPU::h_zs );
  this->d_xs2.SendBuffer( GCAmorphGPU::h_xs2 );
  this->d_ys2.SendBuffer( GCAmorphGPU::h_ys2 );
  this->d_zs2.SendBuffer( GCAmorphGPU::h_zs2 );
  this->d_saved_origx.SendBuffer( GCAmorphGPU::h_saved_origx );
  this->d_saved_origy.SendBuffer( GCAmorphGPU::h_saved_origy );
  this->d_saved_origz.SendBuffer( GCAmorphGPU::h_saved_origz );

  // Wait for the copies to complete
  CUDA_SAFE_CALL( cudaThreadSynchronize() );
  GCAmorphGPU::tSendTransfer.Stop();


  GCAmorphGPU::tSendTot.Stop();

}

// --------------------------------------------

void GCAmorphGPU::RecvAll( GCAM* dst ) const
{
  /*!
  Retrieves all supported data in the given GCAM
  from the GPU.
  This involves a lot of packing data, and hence
  is going to be painfully slow
  */

  GCAmorphGPU::tRecvTot.Start();

  // Check for number of inputs
  if( dst->ninputs != 1 )
  {
    std::cerr << __FUNCTION__
              << ": Must have only one input in the GC1D!"
              << std::endl;
    exit( EXIT_FAILURE );
  }


  // Copy scalars
  dst->exp_k = this->exp_k;
  dst->neg = this->neg;
  std::cerr << __FUNCTION__
            << ": Did not reset gca in dst"
            << std::endl;
  dst->spacing = this->spacing;

  // Extract the dimensions
  const dim3 dims = this->d_rx.GetDims();

  // Allocate page-locked host memory
  GCAmorphGPU::AllocateHost( *this );

  GCAmorphGPU::tRecvTransfer.Start();
  // Fetch the data
  this->d_rx.RecvBuffer( GCAmorphGPU::h_rx );
  this->d_ry.RecvBuffer( GCAmorphGPU::h_ry );
  this->d_rz.RecvBuffer( GCAmorphGPU::h_rz );

  this->d_origx.RecvBuffer( GCAmorphGPU::h_origx );
  this->d_origy.RecvBuffer( GCAmorphGPU::h_origy );
  this->d_origz.RecvBuffer( GCAmorphGPU::h_origz );

  this->d_dx.RecvBuffer( GCAmorphGPU::h_dx );
  this->d_dy.RecvBuffer( GCAmorphGPU::h_dy );
  this->d_dz.RecvBuffer( GCAmorphGPU::h_dz );

  this->d_odx.RecvBuffer( GCAmorphGPU::h_odx );
  this->d_ody.RecvBuffer( GCAmorphGPU::h_ody );
  this->d_odz.RecvBuffer( GCAmorphGPU::h_odz );

  this->d_origArea.RecvBuffer( GCAmorphGPU::h_origArea );
  this->d_origArea1.RecvBuffer( GCAmorphGPU::h_origArea1 );
  this->d_origArea2.RecvBuffer( GCAmorphGPU::h_origArea2 );

  this->d_area.RecvBuffer( GCAmorphGPU::h_area );
  this->d_area1.RecvBuffer( GCAmorphGPU::h_area1 );
  this->d_area2.RecvBuffer( GCAmorphGPU::h_area2 );

  this->d_invalid.RecvBuffer( GCAmorphGPU::h_invalid );
  this->d_status.RecvBuffer( GCAmorphGPU::h_status );
  this->d_label.RecvBuffer( GCAmorphGPU::h_label );
  this->d_labelDist.RecvBuffer( GCAmorphGPU::h_labelDist );

  this->d_mean.RecvBuffer( GCAmorphGPU::h_mean );
  this->d_variance.RecvBuffer( GCAmorphGPU:: h_variance );

  // And the non-saved variables
  this->d_xs.RecvBuffer( GCAmorphGPU::h_xs );
  this->d_ys.RecvBuffer( GCAmorphGPU::h_ys );
  this->d_zs.RecvBuffer( GCAmorphGPU::h_zs );
  this->d_xs2.RecvBuffer( GCAmorphGPU::h_xs2 );
  this->d_ys2.RecvBuffer( GCAmorphGPU::h_ys2 );
  this->d_zs2.RecvBuffer( GCAmorphGPU::h_zs2 );
  this->d_saved_origx.RecvBuffer( GCAmorphGPU::h_saved_origx );
  this->d_saved_origy.RecvBuffer( GCAmorphGPU::h_saved_origy );
  this->d_saved_origz.RecvBuffer( GCAmorphGPU::h_saved_origz );

  CUDA_SAFE_CALL( cudaThreadSynchronize() );
  GCAmorphGPU::tRecvTransfer.Stop();

  GCAmorphGPU::tRecvPack.Start();
  for( unsigned int i=0; i<dims.x; i++ )
  {
    for( unsigned int j=0; j<dims.y; j++ )
    {
      for( unsigned int k=0; k<dims.z; k++ )
      {

        // Get the 1d index (same for all arrays)
        const unsigned int i1d = this->d_rx.Index1D( i, j, k );
        // Get the current node
        GCA_MORPH_NODE* gcamn = &(dst->nodes[i][j][k]);

        gcamn->x = GCAmorphGPU::h_rx[i1d];
        gcamn->y = GCAmorphGPU::h_ry[i1d];
        gcamn->z = GCAmorphGPU::h_rz[i1d];

        gcamn->origx = GCAmorphGPU::h_origx[i1d];
        gcamn->origy = GCAmorphGPU::h_origy[i1d];
        gcamn->origz = GCAmorphGPU::h_origz[i1d];

        gcamn->dx = GCAmorphGPU::h_dx[i1d];
        gcamn->dy = GCAmorphGPU::h_dy[i1d];
        gcamn->dz = GCAmorphGPU::h_dz[i1d];

        gcamn->odx = GCAmorphGPU::h_odx[i1d];
        gcamn->ody = GCAmorphGPU::h_ody[i1d];
        gcamn->odz = GCAmorphGPU::h_odz[i1d];

        gcamn->orig_area = GCAmorphGPU::h_origArea[i1d];
        gcamn->orig_area1 = GCAmorphGPU::h_origArea1[i1d];
        gcamn->orig_area2 = GCAmorphGPU::h_origArea2[i1d];

        gcamn->area = GCAmorphGPU::h_area[i1d];
        gcamn->area1 = GCAmorphGPU::h_area1[i1d];
        gcamn->area2 = GCAmorphGPU::h_area2[i1d];

        gcamn->invalid = GCAmorphGPU::h_invalid[i1d];
        gcamn->label = GCAmorphGPU::h_label[i1d];
        gcamn->label_dist = GCAmorphGPU::h_labelDist[i1d];
        gcamn->status = GCAmorphGPU::h_status[i1d];

        // We now have a quandary... how to test for validity
        if( gcamn->gc != NULL )
        {
          // We know there's only one input from test at the top
          gcamn->gc->means[0] = GCAmorphGPU::h_mean[i1d];
          gcamn->gc->covars[0] = GCAmorphGPU::h_variance[i1d];
        }
        else
        {
          if( GCAmorphGPU::h_variance[i1d] >= 0 )
          {
            std::cerr << __FUNCTION__
                      << ": Host has no GC1D but GPU has valid variance"
                      << std::endl;
            exit( EXIT_FAILURE );
          }
        }

        // And the non-saved members
        gcamn->xs = GCAmorphGPU::h_xs[i1d];
        gcamn->ys = GCAmorphGPU::h_ys[i1d];
        gcamn->zs = GCAmorphGPU::h_zs[i1d];
        gcamn->xs2 = GCAmorphGPU::h_xs2[i1d];
        gcamn->ys2 = GCAmorphGPU::h_ys2[i1d];
        gcamn->zs2 = GCAmorphGPU::h_zs2[i1d];
        gcamn->saved_origx = GCAmorphGPU::h_saved_origx[i1d];
        gcamn->saved_origy = GCAmorphGPU::h_saved_origy[i1d];
        gcamn->saved_origz = GCAmorphGPU::h_saved_origz[i1d];
      }
    }
  }
  GCAmorphGPU::tRecvPack.Stop();

  GCAmorphGPU::tRecvTot.Stop();

}




// --------------------------------------------

const unsigned int kCMPKernelSize = 16;
const unsigned int iCMPGlobalsInvalid = 0;
const unsigned int iCMPGlobalsNeg = 1;

//! Device function to look up displacement vectors
__device__ float3 FetchVector( const unsigned int ix,
                               const unsigned int iy,
                               const unsigned int iz )
{

  float3 r;
  r.x = tex3D( dt_rx, ix+0.5f, iy+0.5f, iz+0.5f );
  r.y = tex3D( dt_ry, ix+0.5f, iy+0.5f, iz+0.5f );
  r.z = tex3D( dt_rz, ix+0.5f, iy+0.5f, iz+0.5f );

  return( r );
}

//! Kernel to perform work of gcamComputeMetricProperties
__global__
void CompMetPropKernel( const VolumeArgGPU<float> origArea,
                        VolumeArgGPU<char> invalid,
                        VolumeArgGPU<float> area,
                        VolumeArgGPU<float> area1,
                        VolumeArgGPU<float> area2,
                        int *globals )
{
  /*!
  This kernel performs the work of gcamComputeMetricProperties.
  For now, it's unoptimised, and may cause a lot of un-necessary
  memory transations
  */
  // Compute co-ordinates
  const unsigned int ix = threadIdx.x + ( blockIdx.x * blockDim.x );
  const unsigned int iy = threadIdx.y + ( blockIdx.y * blockDim.y );

  // Check if in volume
  if( !origArea.InVolume( ix, iy, 0 ) )
  {
    return;
  }

  // Loop over each z slice
  for( unsigned int iz=0; iz< origArea.dims.z; iz++ )
  {

    int neg = 0;
    int num = 0;

    // Check for invalid node
    if( invalid( ix, iy, iz ) == GCAM_POSITION_INVALID )
    {
      atomicAdd( &(globals[iCMPGlobalsInvalid]), 1 );
      continue;
    }

    // Fetch the location of the current voxel
    const float3 r = FetchVector( ix, iy, iz );

    // Zero the 'area'
    area(ix,iy,iz) = 0;

    // Compute Jacobean determinants on the 'right'
    if( (ix<origArea.dims.x-1) &&
        (iy<origArea.dims.y-1) &&
        (iz<origArea.dims.z-1) )
    {


      // Check for validity
      if( (invalid(ix+1,iy,iz) != GCAM_POSITION_INVALID) &&
          (invalid(ix,iy+1,iz) != GCAM_POSITION_INVALID) &&
          (invalid(ix,iy,iz+1) != GCAM_POSITION_INVALID) )
      {

        num++;


        float3 vi = FetchVector(ix+1,iy  ,iz  ) - r;
        float3 vj = FetchVector(ix  ,iy+1,iz  ) - r;
        float3 vk = FetchVector(ix  ,iy  ,iz+1) - r;

        float tmpArea = stp( vj, vk, vi );
        if( tmpArea <= 0 )
        {
          neg = 1;
        }

        area1(ix,iy,iz) = tmpArea;
        area(ix,iy,iz) += tmpArea;

      }
    }
    else
    {
      // Going to 'right' would fall out of the volume
      area1(ix,iy,iz) = 0;
    }


    // Compute Jacobean determinants on the 'left'
    if( (ix>0) && (iy>0) && (iz>0) )
    {

      // Check for validity
      if( (invalid(ix-1,iy,iz) != GCAM_POSITION_INVALID) &&
          (invalid(ix,iy-1,iz) != GCAM_POSITION_INVALID) &&
          (invalid(ix,iy,iz-1) != GCAM_POSITION_INVALID) )
      {
        num++;

        // I think this ordering preserves handedness
        // It's different to that in gcamorph.c
        float3 vi = r - FetchVector(ix-1,iy  ,iz  );
        float3 vj = r - FetchVector(ix  ,iy-1,iz  );
        float3 vk = r - FetchVector(ix  ,iy  ,iz-1);

        float tmpArea = stp( vj, vk, vi );

        if( tmpArea <= 0 )
        {
          neg = 1;
        }

        area2(ix,iy,iz) = tmpArea;
        area(ix,iy,iz) += tmpArea;
      }
    }
    else
    {
      area2(ix,iy,iz) = 0;
    }

    // Check if at least one determinant was computed
    if( num > 0 )
    {
      // area is mean of 'left' and 'right' areas
      area(ix,iy,iz) /= num;
    }
    else
    {
      invalid(ix,iy,iz) = GCAM_AREA_INVALID;
      area(ix,iy,iz) = 0;
    }

    // Keep track of sign changes
    if( (invalid(ix,iy,iz)==GCAM_VALID) &&
        neg &&
        origArea(ix,iy,iz) > 0 )
    {
      atomicAdd( &(globals[iCMPGlobalsNeg]), 1 );
    }

    // Increment invalid counter
    if( invalid(ix,iy,iz) != GCAM_VALID )
    {
      // We need to test again
      atomicAdd( &(globals[iCMPGlobalsInvalid]), 1 );
    }
  }
}

void GCAmorphGPU::ComputeMetricProperties( int& invalid )
{
  /*!
  Routine to duplicate gcamComputeMetricProperties
  from the file gcamorph.c.
  It essentially computes a lot of jacobean determinants
  and sums them up.
  The argument \a invalid is used to return the number of
  invalid locations found, a task performed by the
  global variable \c Ginvalid in gcamorph.c.
  */


  GCAmorphGPU::tCMPtot.Start();

  // Sanity check
  this->CheckIntegrity();

  // Allocate temporary on the device to hold invalid and neg
  int *d_globals;
  CUDA_SAFE_CALL( cudaMalloc( (void**)&d_globals, 2*sizeof(int) ) );
  CUDA_SAFE_CALL( cudaMemset( d_globals, 0, 2*sizeof(int) ) );

  // Get the d_rx, d_ry and d_rz fields bound to textures
  GPU::Classes::CTfactory rxArray( this->d_rx, dt_rx );
  GPU::Classes::CTfactory ryArray( this->d_ry, dt_ry );
  GPU::Classes::CTfactory rzArray( this->d_rz, dt_rz );

  // Run the kernel
  dim3 grid, threads;

  threads.x = threads.y = kCMPKernelSize;
  threads.z = 1;

  grid = this->d_rx.CoverBlocks( kCMPKernelSize );
  grid.z = 1;

  GCAmorphGPU::tCMPcompute.Start();
  CompMetPropKernel<<<grid,threads>>>
  ( this->d_origArea, this->d_invalid,
    this->d_area, this->d_area1, this->d_area2,
    d_globals );
  CUDA_CHECK_ERROR( "CompMetPropKernel failed!\n" );
  GCAmorphGPU::tCMPcompute.Stop();

  // Retrieve global statistics
  int globals[2];
  CUDA_SAFE_CALL( cudaMemcpy( &globals, d_globals,
                              2*sizeof(int),
                              cudaMemcpyDeviceToHost ) );
  invalid = globals[iCMPGlobalsInvalid];
  this->neg = globals[iCMPGlobalsNeg];

  // Release device temporary
  CUDA_SAFE_CALL( cudaFree( d_globals ) );


  GCAmorphGPU::tCMPtot.Stop();
}


// --------------------------------------------



void GCAmorphGPU::ClearGradient( void )
{
  this->d_dx.Zero();
  this->d_dy.Zero();
  this->d_dz.Zero();
}

void GCAmorphGPU::ClearMomentum( void )
{
  this->d_odx.Zero();
  this->d_ody.Zero();
  this->d_odz.Zero();
}


// --------------------------------------------

const unsigned int kApplyGradientKernelSize = 16;

__device__ void FetchDerivs( const unsigned int ix,
                             const unsigned int iy,
                             const unsigned int iz,
                             float& dx, float& dy, float& dz )
{

  const float xLoc = ix+0.5f;
  const float yLoc = iy+0.5f;
  const float zLoc = iz+0.5f;

  dx = tex3D( dt_dx, xLoc, yLoc, zLoc );
  dy = tex3D( dt_dy, xLoc, yLoc, zLoc );
  dz = tex3D( dt_dz, xLoc, yLoc, zLoc );
}

__global__
void ApplyGradientKernel( const VolumeArgGPU<char> invalid,
                          VolumeArgGPU<float> odx,
                          VolumeArgGPU<float> ody,
                          VolumeArgGPU<float> odz,
                          VolumeArgGPU<float> rx,
                          VolumeArgGPU<float> ry,
                          VolumeArgGPU<float> rz,
                          const float dt, const float momentum )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  for( unsigned int iz = 0; iz< invalid.dims.z; iz++ )
  {
    if( invalid.InVolume(ix,iy,iz) )
    {

      if( invalid(ix,iy,iz) == GCAM_POSITION_INVALID )
      {
        continue;
      }

      // Fetch the dx, dy and dz values from textures
      float gcamdx, gcamdy, gcamdz;
      FetchDerivs( ix, iy, iz, gcamdx, gcamdy, gcamdz );

      float ldx, ldy, ldz;

      ldx = gcamdx*dt + odx(ix,iy,iz)*momentum;
      ldy = gcamdy*dt + ody(ix,iy,iz)*momentum;
      ldz = gcamdz*dt + odz(ix,iy,iz)*momentum;

      // Update odx, ody, odz
      odx(ix,iy,iz) = ldx;
      ody(ix,iy,iz) = ldy;
      odz(ix,iy,iz) = ldz;

      // Update x, y z
      rx(ix,iy,iz) += ldx;
      ry(ix,iy,iz) += ldy;
      rz(ix,iy,iz) += ldz;
    }
  }
}

void GCAmorphGPU::ApplyGradient( GCA_MORPH_PARMS *parms )
{

  // Start with a sanity check
  this->CheckIntegrity();

  // Put dx, dy and dz into textures
  GPU::Classes::CTfactory dxArray( this->d_dx, dt_dx );
  GPU::Classes::CTfactory dyArray( this->d_dy, dt_dy );
  GPU::Classes::CTfactory dzArray( this->d_dz, dt_dz );



  // Run the computation
  dim3 grid, threads;

  threads.x = threads.y = kApplyGradientKernelSize;
  threads.z = 1;

  grid = this->d_invalid.CoverBlocks( kApplyGradientKernelSize );
  grid.z = 1;

  ApplyGradientKernel<<<grid,threads>>>
  ( this->d_invalid,
    this->d_odx, this->d_ody, this->d_odz,
    this->d_rx, this->d_ry, this->d_rz,
    parms->dt, parms->momentum );
  CUDA_CHECK_ERROR( "ApplyGradientKernel failed!\n" );



  // Something we can't do yet....
  if (!DZERO(parms->l_area_intensity))
  {
    std::cerr << __FUNCTION__
              << ": gcamCreateNodeLookupTable not implemented!"
              << std::endl;
    exit( EXIT_FAILURE );
  }

}

// --------------------------------------------

const unsigned int kUndoGradientKernelSize = 16;

__global__
void UndoGradientKernel( const VolumeArgGPU<char> invalid,
                         VolumeArgGPU<float> odx,
                         VolumeArgGPU<float> ody,
                         VolumeArgGPU<float> odz,
                         VolumeArgGPU<float> rx,
                         VolumeArgGPU<float> ry,
                         VolumeArgGPU<float> rz )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  for( unsigned int iz = 0; iz< invalid.dims.z; iz++ )
  {
    if( invalid.InVolume(ix,iy,iz) )
    {

      if( invalid(ix,iy,iz) == GCAM_POSITION_INVALID )
      {
        continue;
      }

      float ldx = odx(ix,iy,iz);
      float ldy = ody(ix,iy,iz);
      float ldz = odz(ix,iy,iz);

      // Update odx, ody, odz
      odx(ix,iy,iz) = 0;
      ody(ix,iy,iz) = 0;
      odz(ix,iy,iz) = 0;

      // Update x, y z
      rx(ix,iy,iz) -= ldx;
      ry(ix,iy,iz) -= ldy;
      rz(ix,iy,iz) -= ldz;
    }
  }
}


void GCAmorphGPU::UndoGradient( void )
{

  this->CheckIntegrity();

  // Run the computation
  dim3 grid, threads;

  threads.x = threads.y = kUndoGradientKernelSize;
  threads.z = 1;

  grid = this->d_invalid.CoverBlocks( kUndoGradientKernelSize );
  grid.z = 1;

  UndoGradientKernel<<<grid,threads>>>
  ( this->d_invalid,
    this->d_odx, this->d_ody, this->d_odz,
    this->d_rx, this->d_ry, this->d_rz );
  CUDA_CHECK_ERROR( "UndoGradientKernel failed!\n" );
}

// --------------------------------------------

const unsigned int kAddStatusKernelSize = 16;

__global__
void AddStatusKernel( VolumeArgGPU<int> status, const int addState )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  for( unsigned int iz = 0; iz< status.dims.z; iz++ )
  {
    if( status.InVolume(ix,iy,iz) )
    {
      status(ix,iy,iz) |= addState;
    }
  }
}

void GCAmorphGPU::AddStatus( const int addState )
{

  this->CheckIntegrity();

  // Run the computation
  dim3 grid, threads;

  threads.x = threads.y = kAddStatusKernelSize;
  threads.z = 1;

  grid = this->d_status.CoverBlocks( kAddStatusKernelSize );
  grid.z = 1;

  AddStatusKernel<<<grid,threads>>>( this->d_status, addState );
  CUDA_CHECK_ERROR( "AddStatusKernel failed!" );
}


// --------------------------------------------

const unsigned int kRemoveStatusKernelSize = 16;

__global__
void RemoveStatusKernel( VolumeArgGPU<int> status, const int subtractState )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  const int invState = ~subtractState;

  for( unsigned int iz = 0; iz< status.dims.z; iz++ )
  {
    if( status.InVolume(ix,iy,iz) )
    {
      status(ix,iy,iz) &= invState;
    }
  }
}

void GCAmorphGPU::RemoveStatus( const int subtractState )
{

  this->CheckIntegrity();

  // Run the computation
  dim3 grid, threads;

  threads.x = threads.y = kRemoveStatusKernelSize;
  threads.z = 1;

  grid = this->d_status.CoverBlocks( kRemoveStatusKernelSize );
  grid.z = 1;

  RemoveStatusKernel<<<grid,threads>>>( this->d_status, subtractState );
  CUDA_CHECK_ERROR( "AddStatusKernel failed!" );
}

// --------------------------------------------

void GCAmorphGPU::ResetLabelNodeStatus( void )
{
  this->RemoveStatus( GCAM_LABEL_NODE );
  this->RemoveStatus( GCAM_IGNORE_LIKELIHOOD );
}


// --------------------------------------------

void GCAmorphGPU::SmoothGradient( const int nAvgs )
{
  /*!
  A re-implementation of gcamSmoothGradient for
  the GPU.
  This is going to get very, very messy.
  Almost as messy as the CPU routine... at least
  we already have a structure of arrays for the
  GCAmorph....
  */

  GPU::Algorithms::MRIconvolve myConvolution;

  if( nAvgs <= 0 )
  {
    return;
  }

  GCAmorphGPU::tSmoothGradient.Start();


  this->CheckIntegrity();
  const dim3 myDims = this->d_dx.GetDims();

  // Set up the kernel
  MRI *mri_kernel;

  mri_kernel = MRIgaussian1d(sqrt((float)nAvgs*2/M_PI), 0 );
  const int klen = mri_kernel->width;

  myConvolution.BindKernel( &MRIFvox(mri_kernel, 0, 0, 0), klen );

  MRIframeGPU<float> d_tmp1, d_tmp2;

  d_tmp1.Allocate( myDims );
  d_tmp2.Allocate( myDims );

  /*
  And now boys and girls, let's blow type safety to smithereens.
  We are going to coerce the VolumeGPU fields of the GCAmorph
  structure into MRIframeGPU types, so we can use the canned
  convolution routines.
  What we should do is make convolutions available to the
  VolumeGPU base class.
  However, that would require quite a bit of coding.
  */
  MRIframeGPU<float> *curr;
  curr = reinterpret_cast< MRIframeGPU<float>* >(&(this->d_dx));

  // Do some convolving
  myConvolution.RunGPU1D( *curr, d_tmp1, MRI_WIDTH );
  myConvolution.RunGPU1D( d_tmp1, d_tmp2, MRI_HEIGHT );
  myConvolution.RunGPU1D( d_tmp2, *curr, MRI_DEPTH );

  // Move on to dy
  curr = reinterpret_cast< MRIframeGPU<float>* >(&(this->d_dy));

  // Do some convolving
  myConvolution.RunGPU1D( *curr, d_tmp1, MRI_WIDTH );
  myConvolution.RunGPU1D( d_tmp1, d_tmp2, MRI_HEIGHT );
  myConvolution.RunGPU1D( d_tmp2, *curr, MRI_DEPTH );

  // And finally dz
  curr = reinterpret_cast< MRIframeGPU<float>* >(&(this->d_dz));

  // Do some convolving
  myConvolution.RunGPU1D( *curr, d_tmp1, MRI_WIDTH );
  myConvolution.RunGPU1D( d_tmp1, d_tmp2, MRI_HEIGHT );
  myConvolution.RunGPU1D( d_tmp2, *curr, MRI_DEPTH );

  // Release things
  myConvolution.UnbindKernel();

  MRIfree( &mri_kernel );


  GCAmorphGPU::tSmoothGradient.Stop();
}


// -------------------------------------------------------------

const unsigned int kWriteWarpToVecVolKernelSize = 16;

__global__
void WriteWarpToVecVolKernel( VecVolArgGPU vv,
                              const VolumeArgGPU<float> x,
                              const VolumeArgGPU<float> y,
                              const VolumeArgGPU<float> z,
                              const VolumeArgGPU<float> origx,
                              const VolumeArgGPU<float> origy,
                              const VolumeArgGPU<float> origz )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  for( unsigned int iz = 0; iz<x.dims.z; iz++ )
  {
    if( x.InVolume(ix,iy,iz) )
    {
      float3 res;
      res.x = x(ix,iy,iz) - origx(ix,iy,iz);
      res.y = y(ix,iy,iz) - origy(ix,iy,iz);
      res.z = z(ix,iy,iz) - origz(ix,iy,iz);

      vv.Set( res, ix, iy, iz );
    }
  }

}

void GCAmorphGPU::WriteWarpToVecVol( VecVolGPU& vecVol ) const
{
  /*!
    This is a reimplementation of GCAMwriteWarpToMRI.
    We don't have to worry about the transforms or the sampling
    in this case, since we directly copy (with slight modification)
    the rx, ry and rz fields into the VecVolGPU
  */
  GCAmorphGPU::tWriteWarp.Start();

  this->CheckIntegrity();

  // Allocate space
  vecVol.Allocate( this->d_rx.GetDims() );

  // Run the computation
  dim3 grid, threads;

  threads.x = threads.y = kWriteWarpToVecVolKernelSize;
  threads.z = 1;

  grid = this->d_rx.CoverBlocks( kWriteWarpToVecVolKernelSize );
  grid.z = 1;

  WriteWarpToVecVolKernel<<<grid,threads>>>( vecVol,
      this->d_rx,
      this->d_ry,
      this->d_rz,
      this->d_origx,
      this->d_origy,
      this->d_origz );
  CUDA_CHECK_ERROR( "WriteWarpToVecVolKernel failed!" );
  GCAmorphGPU::tWriteWarp.Stop();
}





const unsigned int kReadWarpFromVecVolKernelSize = 16;

__global__
void ReadWarpFromVecVolKernel( VolumeArgGPU<float> x,
                               VolumeArgGPU<float> y,
                               VolumeArgGPU<float> z,
                               const VolumeArgGPU<float> origx,
                               const VolumeArgGPU<float> origy,
                               const VolumeArgGPU<float> origz,
                               const VecVolArgGPU vv )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  for( unsigned int iz = 0; iz<x.dims.z; iz++ )
  {
    if( x.InVolume(ix,iy,iz) )
    {
      const float3 vec = vv(ix,iy,iz);
      x(ix,iy,iz) = origx(ix,iy,iz) + vec.x;
      y(ix,iy,iz) = origy(ix,iy,iz) + vec.y;
      z(ix,iy,iz) = origz(ix,iy,iz) + vec.z;
    }
  }
}


void GCAmorphGPU::ReadWarpFromVecVol( const VecVolGPU& vecVol )
{
  /*
    This reimplements GCAMreadWarpFromMRI.
    But again, we don't have to worry about 'spacing' and sampling
  */
  GCAmorphGPU::tReadWarp.Start();

  this->CheckIntegrity();
  if( vecVol.GetDims() != this->d_rx.GetDims() )
  {
    std::cerr << __FUNCTION__
              << ": Volume size mismatch"
              << std::endl;
    abort();
  }

  // Run the computation
  dim3 grid, threads;
  threads.x = threads.y = kReadWarpFromVecVolKernelSize;
  threads.z = 1;

  grid = this->d_rx.CoverBlocks( kReadWarpFromVecVolKernelSize );
  grid.z = 1;

  ReadWarpFromVecVolKernel<<<grid,threads>>>( this->d_rx,
      this->d_ry,
      this->d_rz,
      this->d_origx,
      this->d_origy,
      this->d_origz,
      vecVol );
  CUDA_CHECK_ERROR( "ReadWarpFromVecVolKernel failed!" );

  GCAmorphGPU::tReadWarp.Stop();
}




// --------------------------------------------

const unsigned int kRemoveSingularitiesKernelSize = 16;

__device__
int ClampRange( const int val, unsigned int maxVal )
{
  int res;
  if( val < 0 )
  {
    res = 0;
  }
  else if( val >= static_cast<int>(maxVal) )
  {
    res = maxVal-1;
  }
  else
  {
    res = val;
  }

  return( res );
}


//! Mean of vectors around location
__device__
float3 VoxelMean( const VecVolArgGPU volume,
                  const int x0,
                  const int y0,
                  const int z0,
                  const int wsize )
{
  /*!
    Implementation of MRIvoxelMean for the
    RemoveSingularities kernel
  */
  float3 res = make_float3(0,0,0);

  const int whalf = wsize / 2;
  const int xmin = max( 0, x0-whalf );
  const int xmax = min( volume.dims.x-1, x0+whalf );
  const int ymin = max( 0, y0-whalf );
  const int ymax = min( volume.dims.y-1, y0+whalf );
  const int zmin = max( 0, z0-whalf );
  const int zmax = min( volume.dims.z-1, z0+whalf );

  const int npix = (zmax-zmin+1) * (ymax-ymin+1) * (xmax-xmin+1);

  for( int z=zmin; z<=zmax; z++ )
  {
    for( int y=ymin; y<=ymax; y++ )
    {
      for( int x=xmin; x<=xmax; x++ )
      {
        res += volume(x,y,z);
      }
    }
  }

  if( npix >= 0 )
  {
    res /= static_cast<float>(npix);
  }
  else
  {
    res = make_float3(0,0,0);
  }

  return( res );
}


__global__
void RemoveSingularitiesKernel( VecVolArgGPU warp,
                                const VecVolArgGPU tmpWarp,
                                const VolumeArgGPU<float> area1,
                                const VolumeArgGPU<float> area2,
                                const VolumeArgGPU<char> invalid,
                                const int wsize,
                                const int nbhd )
{
  const unsigned int bx = ( blockIdx.x * blockDim.x );
  const unsigned int by = ( blockIdx.y * blockDim.y );
  const unsigned int ix = threadIdx.x + bx;
  const unsigned int iy = threadIdx.y + by;

  for( unsigned int iz = 0; iz< invalid.dims.z; iz++ )
  {
    if( invalid.InVolume(ix,iy,iz) )
    {

      const bool neg1 = ( area1(ix,iy,iz) < 0 );
      const bool neg2 = ( area2(ix,iy,iz) < 0 );
      const bool valid = ( invalid(ix,iy,iz) == GCAM_VALID );

      if( ( neg1 || neg2 ) && valid )
      {

        const dim3 dims = invalid.dims;

        // Loop over neighbourhood
        for( int zk=-nbhd; zk<=nbhd; zk++ )
        {

          const int zv = iz+zk;
          if( (zv<0) || (zv>=dims.z) )
          {
            continue;
          }

          for( int yk=-nbhd; yk<nbhd; yk++ )
          {

            const int yv = iy+yk;
            if( (yv<0) || (yv>=dims.y) )
            {
              continue;
            }

            for( int xk=-nbhd; xk<=nbhd; xk++ )
            {

              const int xv = ix+xk;
              if( (xv<0) || (xv>=dims.x) )
              {
                continue;
              }

              // Neighbouring voxel in the volume
              float3 sv = VoxelMean( tmpWarp, xv, yv, zv, wsize );

              // This is potentially a race condition (I think....)
              warp.Set( sv, xv, yv, zv );
            }
          }
        }


      }

    }
  }
}

void GCAmorphGPU::RemoveSingularities( void )
{
  /*!
    An implementation of GCAMremoveSingularitiesAndReadWarpFromMRI
    for the GPU.
    We drop the mri_warp, and handle it internally.
  */
  GCAmorphGPU::tRStot.Start();

  int invalid, wsize;

  VecVolGPU warp, tmpWarp;

  this->CheckIntegrity();

  this->WriteWarpToVecVol( warp );

  wsize = 3;

  // See if anything has to be done
  this->ReadWarpFromVecVol( warp );
  this->ComputeMetricProperties( invalid );
  if( this->neg == 0 )
  {
    return;
  }

  int iter = 0;
  int noprogress = 0;
  const int max_iter = 500;
  const int max_noprogress = 4;
  int min_neg = this->neg;
  int max_nbhd = 3;

  int nbhd = 1;
  if( this->spacing-1 > nbhd )
  {
    nbhd = this->spacing-1;
  }

  printf("iter %d, gcam->neg = %d\n", iter, this->neg );


  int last_neg;

  // Main loop
  do
  {
    tmpWarp.Copy( warp );
    last_neg = this->neg;

    // GPU smoothing
    dim3 grid, threads;
    threads.x = threads.y = kRemoveSingularitiesKernelSize;
    threads.z = 1;

    grid = this->d_rx.CoverBlocks( kRemoveSingularitiesKernelSize );
    grid.z = 1;

    GCAmorphGPU::tRScompute.Start();
    RemoveSingularitiesKernel<<<grid,threads>>>( warp, tmpWarp,
        this->d_area1, this->d_area2,
        this->d_invalid,
        wsize,
        nbhd );
    CUDA_CHECK_ERROR( "RemoveSingularitiesKernel failed!" );
    GCAmorphGPU::tRScompute.Stop();

    // Check for negatives
    this->ReadWarpFromVecVol( warp );
    this->ComputeMetricProperties( invalid );

    printf("iter %d, gcam->neg = %d, nbhd=%d\n", iter+1, this->neg, nbhd) ;

    // Determine next step
    if( this->neg >= min_neg )
    {
      if( noprogress++ >= max_noprogress )
      {
        nbhd++;
        if( nbhd > max_nbhd )
        {
          nbhd = 1;
          max_nbhd++;
        }

        noprogress = 0;
      }
    }
    else
    {
      noprogress = 0;
      min_neg = this->neg;
    }

  }
  while( (this->neg>0) && ( (++iter < max_iter) || (this->neg < last_neg ) ) );

  printf("after %d iterations, nbhd size=%d, neg = %d\n", iter, nbhd, this->neg) ;

  GCAmorphGPU::tRStot.Stop();
}




// --------------------------------------------

void GCAmorphGPU::CopyNodePositions( const int from, const int to )
{
  /*!
  This is a reimplementation using the GPU copies
  of GCAMcopyNodePositions.
  Really, I think that the GCAM should be torn apart,
  and this would happen as a copy between two GCAMs
  */
  this->CheckIntegrity();

  switch( from )
  {

  case ORIGINAL_POSITIONS:
    switch( to )
    {

    case SAVED_ORIGINAL_POSITIONS:
      this->d_saved_origx.Copy( this->d_origx );
      this->d_saved_origy.Copy( this->d_origy );
      this->d_saved_origz.Copy( this->d_origz );
      break;

    case SAVED_POSITIONS:
      this->d_xs.Copy( this->d_origx );
      this->d_ys.Copy( this->d_origy );
      this->d_zs.Copy( this->d_origz );
      break;

    case CURRENT_POSITIONS:
      this->d_rx.Copy( this->d_origx );
      this->d_ry.Copy( this->d_origy );
      this->d_rz.Copy( this->d_origz );
      break;

    default:
      std::cerr << __FUNCTION__
                << ": Unrecognised to = " << to << std::endl;
      abort();
    }
    break;

    // -----------------------

  case SAVED_ORIGINAL_POSITIONS:
    switch( to )
    {

    case SAVED_POSITIONS:
      this->d_xs.Copy( this->d_saved_origx );
      this->d_ys.Copy( this->d_saved_origy );
      this->d_zs.Copy( this->d_saved_origz );
      break;

    case CURRENT_POSITIONS:
      this->d_rx.Copy( this->d_saved_origx );
      this->d_ry.Copy( this->d_saved_origy );
      this->d_rz.Copy( this->d_saved_origz );
      break;

    case ORIGINAL_POSITIONS:
      this->d_origx.Copy( this->d_saved_origx );
      this->d_origy.Copy( this->d_saved_origy );
      this->d_origz.Copy( this->d_saved_origz );
      break;

    default:
      std::cerr << __FUNCTION__
                << ": Unrecognised to = " << to << std::endl;
      abort();
    }
    break;

    // -----------------------

  case SAVED_POSITIONS:
    switch( to )
    {

    case ORIGINAL_POSITIONS:
      this->d_origx.Copy( this->d_xs );
      this->d_origy.Copy( this->d_ys );
      this->d_origz.Copy( this->d_zs );
      break;

    case CURRENT_POSITIONS:
      this->d_rx.Copy( this->d_xs );
      this->d_ry.Copy( this->d_ys );
      this->d_rz.Copy( this->d_zs );
      break;

    case SAVED_ORIGINAL_POSITIONS:
      this->d_saved_origx.Copy( this->d_xs );
      this->d_saved_origy.Copy( this->d_ys );
      this->d_saved_origz.Copy( this->d_zs );
      break;

    default:
      std::cerr << __FUNCTION__
                << ": Unrecognised to = " << to << std::endl;
      abort();
    }
    break;


    // -----------------------

  case SAVED2_POSITIONS:
    switch( to )
    {

    case ORIGINAL_POSITIONS:
      this->d_origx.Copy( this->d_xs2 );
      this->d_origy.Copy( this->d_ys2 );
      this->d_origz.Copy( this->d_zs2 );
      break;

    case CURRENT_POSITIONS:
      this->d_rx.Copy( this->d_xs2 );
      this->d_ry.Copy( this->d_ys2 );
      this->d_rz.Copy( this->d_zs2 );
      break;

    case SAVED_ORIGINAL_POSITIONS:
      this->d_saved_origx.Copy( this->d_xs2 );
      this->d_saved_origy.Copy( this->d_ys2 );
      this->d_saved_origz.Copy( this->d_zs2 );
      break;

    default:
      std::cerr << __FUNCTION__
                << ": Unrecognised to = " << to << std::endl;
      abort();
    }
    break;

    // -----------------------

  case CURRENT_POSITIONS:
    switch( to )
    {

    case ORIGINAL_POSITIONS:
      this->d_origx.Copy( this->d_rx );
      this->d_origy.Copy( this->d_ry );
      this->d_origz.Copy( this->d_rz );
      break;

    case SAVED_POSITIONS:
      this->d_xs.Copy( this->d_rx );
      this->d_ys.Copy( this->d_ry );
      this->d_zs.Copy( this->d_rz );
      break;

    case SAVED2_POSITIONS:
      this->d_xs2.Copy( this->d_rx );
      this->d_ys2.Copy( this->d_ry );
      this->d_zs2.Copy( this->d_rz );
      break;

    case SAVED_ORIGINAL_POSITIONS:
      this->d_saved_origx.Copy( this->d_rx );
      this->d_saved_origy.Copy( this->d_ry );
      this->d_saved_origz.Copy( this->d_rz );
      break;

    default:
      std::cerr << __FUNCTION__
                << ": Unrecognised to = " << to << std::endl;
      abort();
    }
    break;

    // -----------------------

  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised from = " << from << std::endl;
    abort();
  }

}


// ----------------------------------------------------
void GCAmorphGPU::ShowTimings( void )
{
#ifdef CUDA_SHOW_TIMINGS
  std::cout << "==================================" << std::endl;
  std::cout << "GCAmorphGPU timers" << std::endl;
  std::cout << "------------------" << std::endl;
#ifndef CUDA_FORCE_SYNC
  std::cout << "WARNING: CUDA_FORCE_SYNC not #defined" << std::endl;
  std::cout << "Timings may not be accurate" << std::endl;
#endif
  std::cout << std::endl;

  std::cout << "Send:" << std::endl;
  std::cout << "      Pack : " << GCAmorphGPU::tSendPack << std::endl;
  std::cout << "  Transfer : " << GCAmorphGPU::tSendTransfer << std::endl;
  std::cout << "Total      : " << GCAmorphGPU::tSendTot << std::endl;
  std::cout << std::endl;

  std::cout << "Recv:" << std::endl;
  std::cout << "      Pack : " << GCAmorphGPU::tRecvPack << std::endl;
  std::cout << "  Transfer : " << GCAmorphGPU::tRecvTransfer << std::endl;
  std::cout << "Total      : " << GCAmorphGPU::tRecvTot << std::endl;
  std::cout << std::endl;

  std::cout << "Host Memory:" << std::endl;
  std::cout << "     Alloc : " << GCAmorphGPU::tHostAlloc << std::endl;
  std::cout << "   Release : " << GCAmorphGPU::tHostRelease << std::endl;
  std::cout << " Randomise : " << GCAmorphGPU::tHostRandomise << std::endl;
  std::cout << std::endl;

  std::cout << "Compute Metric Properties:" << std::endl;
  std::cout << "   Compute : " << GCAmorphGPU::tCMPcompute << std::endl;
  std::cout << "Total      : " << GCAmorphGPU::tCMPtot << std::endl;
  std::cout << std::endl;

  std::cout << "SmoothGradient:" << std::endl;
  std::cout << "Total         : " << GCAmorphGPU::tSmoothGradient << std::endl;
  std::cout << std::endl;

  std::cout << "Remove Singularities:" << std::endl;
  std::cout << "   Compute : " << GCAmorphGPU::tRScompute << std::endl;
  std::cout << "Total      : " << GCAmorphGPU::tRStot << std::endl;
  std::cout << std::endl;


  std::cout << "WriteWarp   : " << GCAmorphGPU::tWriteWarp << std::endl;
  std::cout << "ReadWarp    : " << GCAmorphGPU::tReadWarp << std::endl;

  std::cout << "==================================" << std::endl;
#endif
}



// Define static members
SciGPU::Utilities::Chronometer GCAmorphGPU::tSendTot;
SciGPU::Utilities::Chronometer GCAmorphGPU::tSendPack;
SciGPU::Utilities::Chronometer GCAmorphGPU::tSendTransfer;
SciGPU::Utilities::Chronometer GCAmorphGPU::tRecvTot;
SciGPU::Utilities::Chronometer GCAmorphGPU::tRecvPack;
SciGPU::Utilities::Chronometer GCAmorphGPU::tRecvTransfer;
SciGPU::Utilities::Chronometer GCAmorphGPU::tHostAlloc;
SciGPU::Utilities::Chronometer GCAmorphGPU::tHostRelease;
SciGPU::Utilities::Chronometer GCAmorphGPU::tHostRandomise;
SciGPU::Utilities::Chronometer GCAmorphGPU::tCMPtot;
SciGPU::Utilities::Chronometer GCAmorphGPU::tCMPcompute;
SciGPU::Utilities::Chronometer GCAmorphGPU::tSmoothGradient;
SciGPU::Utilities::Chronometer GCAmorphGPU::tWriteWarp;
SciGPU::Utilities::Chronometer GCAmorphGPU::tReadWarp;
SciGPU::Utilities::Chronometer GCAmorphGPU::tRStot;
SciGPU::Utilities::Chronometer GCAmorphGPU::tRScompute;


dim3 GCAmorphGPU::hostDims = make_uint3(0,0,0);
float *GCAmorphGPU::h_rx, *GCAmorphGPU::h_ry, *GCAmorphGPU::h_rz;
float *GCAmorphGPU::h_origx, *GCAmorphGPU::h_origy, *GCAmorphGPU::h_origz;
float *GCAmorphGPU::h_dx, *GCAmorphGPU::h_dy, *GCAmorphGPU::h_dz;
float *GCAmorphGPU::h_odx, *GCAmorphGPU::h_ody, *GCAmorphGPU::h_odz;
float *GCAmorphGPU::h_origArea, *GCAmorphGPU::h_origArea1, *GCAmorphGPU::h_origArea2;
float *GCAmorphGPU::h_area, *GCAmorphGPU::h_area1, *GCAmorphGPU::h_area2;
char *GCAmorphGPU::h_invalid;
int *GCAmorphGPU::h_label, *GCAmorphGPU::h_status;
float *GCAmorphGPU::h_labelDist;
float *GCAmorphGPU::h_mean;
float *GCAmorphGPU::h_variance;
float *GCAmorphGPU::h_xs, *GCAmorphGPU::h_ys, *GCAmorphGPU::h_zs;
float *GCAmorphGPU::h_xs2, *GCAmorphGPU::h_ys2, *GCAmorphGPU::h_zs2;
float *GCAmorphGPU::h_saved_origx;
float *GCAmorphGPU::h_saved_origy;
float *GCAmorphGPU::h_saved_origz;


void GCAmorphGPU::AllocateHost( const GCAmorphGPU& gcam )
{

  // Check integrity
  gcam.CheckIntegrity();

  // Check if current allocation OK
  const dim3 gcamDims = gcam.d_rx.GetDims();
  const size_t reqSize = gcamDims.x * gcamDims.y * gcamDims.z;
  size_t currSize = GCAmorphGPU::hostDims.x * GCAmorphGPU::hostDims.y * GCAmorphGPU::hostDims.z;

  if( reqSize <= currSize )
  {
    return;
  }

  std::cerr << __FUNCTION__
            << ": Warning - not thread safe!" << std::endl;

  // Get rid of the old allocation
  GCAmorphGPU::ReleaseHost();

  GCAmorphGPU::tHostAlloc.Start();
  // Set dimensions
  GCAmorphGPU::hostDims = gcam.d_rx.GetDims();

  // Do the allocations
  GCAmorphGPU::h_rx = gcam.d_rx.AllocateHostBuffer();
  GCAmorphGPU::h_ry = gcam.d_ry.AllocateHostBuffer();
  GCAmorphGPU::h_rz = gcam.d_rz.AllocateHostBuffer();

  GCAmorphGPU::h_origx = gcam.d_origx.AllocateHostBuffer();
  GCAmorphGPU::h_origy = gcam.d_origy.AllocateHostBuffer();
  GCAmorphGPU::h_origz = gcam.d_origz.AllocateHostBuffer();

  GCAmorphGPU::h_dx = gcam.d_dx.AllocateHostBuffer();
  GCAmorphGPU::h_dy = gcam.d_dy.AllocateHostBuffer();
  GCAmorphGPU::h_dz = gcam.d_dz.AllocateHostBuffer();

  GCAmorphGPU::h_odx = gcam.d_odx.AllocateHostBuffer();
  GCAmorphGPU::h_ody = gcam.d_ody.AllocateHostBuffer();
  GCAmorphGPU::h_odz = gcam.d_odz.AllocateHostBuffer();

  GCAmorphGPU::h_origArea = gcam.d_origArea.AllocateHostBuffer();
  GCAmorphGPU::h_origArea1 = gcam.d_origArea1.AllocateHostBuffer();
  GCAmorphGPU::h_origArea2 = gcam.d_origArea2.AllocateHostBuffer();

  GCAmorphGPU::h_area = gcam.d_area.AllocateHostBuffer();
  GCAmorphGPU::h_area1 = gcam.d_area1.AllocateHostBuffer();
  GCAmorphGPU::h_area2 = gcam.d_area2.AllocateHostBuffer();

  GCAmorphGPU::h_invalid = gcam.d_invalid.AllocateHostBuffer();
  GCAmorphGPU::h_status = gcam.d_status.AllocateHostBuffer();
  GCAmorphGPU::h_label = gcam.d_label.AllocateHostBuffer();
  GCAmorphGPU::h_labelDist = gcam.d_labelDist.AllocateHostBuffer();

  GCAmorphGPU::h_mean = gcam.d_mean.AllocateHostBuffer();
  GCAmorphGPU::h_variance = gcam.d_variance.AllocateHostBuffer();

  // And the non-saved volumes
  GCAmorphGPU::h_xs = gcam.d_xs.AllocateHostBuffer();
  GCAmorphGPU::h_ys = gcam.d_ys.AllocateHostBuffer();
  GCAmorphGPU::h_zs = gcam.d_zs.AllocateHostBuffer();
  GCAmorphGPU::h_xs2 = gcam.d_xs2.AllocateHostBuffer();
  GCAmorphGPU::h_ys2 = gcam.d_ys2.AllocateHostBuffer();
  GCAmorphGPU::h_zs2 = gcam.d_zs2.AllocateHostBuffer();
  GCAmorphGPU::h_saved_origx = gcam.d_saved_origx.AllocateHostBuffer();
  GCAmorphGPU::h_saved_origy = gcam.d_saved_origy.AllocateHostBuffer();
  GCAmorphGPU::h_saved_origz = gcam.d_saved_origz.AllocateHostBuffer();


  GCAmorphGPU::tHostAlloc.Stop();

}



void GCAmorphGPU::ReleaseHost( void )
{

  // Sanity check
  if( GCAmorphGPU::hostDims == make_uint3(0,0,0) )
  {
    return;
  }

  std::cerr << __FUNCTION__
            << ": Warning - not thread safe!" << std::endl;

  GCAmorphGPU::tHostRelease.Start();

  GCAmorphGPU::hostDims = make_uint3(0,0,0);

  // Release page-locked host memory
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_rx ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_ry ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_rz ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origx ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origy ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origz ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_dx ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_dy ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_dz ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_odx ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_ody ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_odz ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origArea ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origArea1 ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_origArea2 ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_area ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_area1 ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_area2 ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_invalid ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_status ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_label ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_labelDist ) );

  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_mean ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_variance ) );

  // The non-saved members
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_xs ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_ys ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_zs ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_xs2 ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_ys2 ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_zs2 ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_saved_origx ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_saved_origy ) );
  CUDA_SAFE_CALL( cudaFreeHost( GCAmorphGPU::h_saved_origz ) );

  GCAmorphGPU::tHostRelease.Stop();
}




template<typename T>
void RandomArray( T* arr, const size_t nVals )
{

  for( unsigned int i=0; i<nVals; i++ )
  {
    char randVal = std::rand() % std::numeric_limits<char>::max();
    arr[i] = static_cast<T>(randVal);
  }

}


void GCAmorphGPU::RandomiseHost( void )
{

  // Sanity check
  if( GCAmorphGPU::hostDims == make_uint3(0,0,0) )
  {
    return;
  }

  std::cerr << __FUNCTION__
            << ": Warning - not thread safe!" << std::endl;

  GCAmorphGPU::tHostRandomise.Start();

  size_t currSize = GCAmorphGPU::hostDims.x *
                    GCAmorphGPU::hostDims.y * GCAmorphGPU::hostDims.z;

  RandomArray( GCAmorphGPU::h_rx, currSize );
  RandomArray( GCAmorphGPU::h_ry, currSize );
  RandomArray( GCAmorphGPU::h_rz, currSize );


  RandomArray( GCAmorphGPU::h_origx, currSize );
  RandomArray( GCAmorphGPU::h_origy, currSize );
  RandomArray( GCAmorphGPU::h_origz, currSize );

  RandomArray( GCAmorphGPU::h_dx, currSize );
  RandomArray( GCAmorphGPU::h_dy, currSize );
  RandomArray( GCAmorphGPU::h_dz, currSize );

  RandomArray( GCAmorphGPU::h_odx, currSize );
  RandomArray( GCAmorphGPU::h_ody, currSize );
  RandomArray( GCAmorphGPU::h_odz, currSize );

  RandomArray( GCAmorphGPU::h_origArea, currSize );
  RandomArray( GCAmorphGPU::h_origArea1, currSize );
  RandomArray( GCAmorphGPU::h_origArea2, currSize );

  RandomArray( GCAmorphGPU::h_area, currSize );
  RandomArray( GCAmorphGPU::h_area1, currSize );
  RandomArray( GCAmorphGPU::h_area2, currSize );

  RandomArray( GCAmorphGPU::h_invalid, currSize );
  RandomArray( GCAmorphGPU::h_status, currSize );
  RandomArray( GCAmorphGPU::h_label, currSize );
  RandomArray( GCAmorphGPU::h_labelDist, currSize );

  RandomArray( GCAmorphGPU::h_mean, currSize );
  RandomArray( GCAmorphGPU::h_variance, currSize );

  // And the non-saved....
  RandomArray( GCAmorphGPU::h_xs, currSize );
  RandomArray( GCAmorphGPU::h_ys, currSize );
  RandomArray( GCAmorphGPU::h_zs, currSize );
  RandomArray( GCAmorphGPU::h_xs2, currSize );
  RandomArray( GCAmorphGPU::h_ys2, currSize );
  RandomArray( GCAmorphGPU::h_zs2, currSize );
  RandomArray( GCAmorphGPU::h_saved_origx, currSize );
  RandomArray( GCAmorphGPU::h_saved_origy, currSize );
  RandomArray( GCAmorphGPU::h_saved_origz, currSize );

  GCAmorphGPU::tHostRandomise.Stop();
}

}
}


void gcamClearGradientGPU( GCA_MORPH* gcam )
{
  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.ClearGradient();
  gcamGPU.RecvAll( gcam );

}

void gcamClearMomentumGPU( GCA_MORPH* gcam )
{
  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.ClearMomentum();
  gcamGPU.RecvAll( gcam );

}


void gcamComputeMetricPropertiesGPU( GCA_MORPH* gcam,
                                     int *invalid )
{
  /*!
    This is a wrapper around the CUDA implementation
    of gcamComputeMetricProperties
  */

  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.ComputeMetricProperties( *invalid );
  gcamGPU.RecvAll( gcam );

}


void gcamApplyGradientGPU( GCA_MORPH *gcam, GCA_MORPH_PARMS *parms )
{

  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.ApplyGradient( parms );
  gcamGPU.RecvAll( gcam );
}


void gcamUndoGradientGPU( GCA_MORPH *gcam )
{

  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.UndoGradient();
  gcamGPU.RecvAll( gcam );
}


void gcamAddStatusGPU( GCA_MORPH *gcam, const int statusFlags )
{

  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.AddStatus( statusFlags );
  gcamGPU.RecvAll( gcam );
}


void gcamRemoveStatusGPU( GCA_MORPH *gcam, const int statusFlags )
{

  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.RemoveStatus( statusFlags );
  gcamGPU.RecvAll( gcam );
}


void gcamSmoothGradientGPU( GCA_MORPH *gcam, int navgs )
{
  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.SmoothGradient( navgs );
  gcamGPU.RecvAll( gcam );
}

void GCAMcopyNodePositionsGPU( GCA_MORPH *gcam,
                               const int from,
                               const int to )
{
  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.CopyNodePositions( from, to );
  gcamGPU.RecvAll( gcam );
}



void GCAMremoveSingularitiesGPU( GCA_MORPH *gcam )
{
  GPU::Classes::GCAmorphGPU gcamGPU;

  gcamGPU.SendAll( gcam );
  gcamGPU.RemoveSingularities();
  gcamGPU.RecvAll( gcam );
}


#endif
