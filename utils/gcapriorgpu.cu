/**
 * @file  gcapriorgpu.hpp
 * @brief Class to hold a volume of GCA priors in linear memory on the GPU
 *
 */
/*
 * Original Authors: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:07 $
 *    $Revision: 1.2.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include "cudacheck.h"

#include "gcapriorgpu.hpp"


namespace GPU
{
namespace Classes
{

void GCApriorGPU::Allocate( const long long nxDim,
                            const long long nyDim,
                            const long long nzDim,
                            const size_t num4D )
{
  // Get rid of old allocation
  this->Release();

  // Copy sizes
  this->xDim = nxDim;
  this->yDim = nyDim;
  this->zDim = nzDim;

  this->n4D = num4D;

  // Do the allocation
  const size_t nVoxels = this->xDim * this->yDim * this->zDim;

  // The offset array
  CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_offsets4D),
                              (nVoxels+1)*sizeof(size_t) ) );

  // Space for maxLabels
  CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_maxLabels),
                              nVoxels*sizeof(short) ) );

  // Space for the labels
  CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_labels),
                              this->n4D*sizeof(unsigned short) ) );

  // Space for the priors
  CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_priors),
                              this->n4D*sizeof(float) ) );

  // Space for the total_training
  CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_totTraining),
                              nVoxels*sizeof(int) ) );
}


// --------------------

void GCApriorGPU::Release( void )
{

  if( this->xDim != 0 )
  {
    // Release offset array
    CUDA_SAFE_CALL( hipFree( this->d_offsets4D ) );
    this->d_offsets4D = NULL;

    // Release 3D arrays
    CUDA_SAFE_CALL( hipFree( this->d_maxLabels ) );
    this->d_maxLabels = NULL;
    CUDA_SAFE_CALL( hipFree( this->d_totTraining ) );
    this->d_totTraining = NULL;

    // Release 4D arrays
    CUDA_SAFE_CALL( hipFree( this->d_labels ) );
    this->d_labels = NULL;
    CUDA_SAFE_CALL( hipFree( this->d_priors ) );
    this->d_priors = NULL;

    // Zero sizes
    this->xDim = 0;
    this->yDim = 0;
    this->zDim = 0;
    this->n4D = 0;

  }
}


// --------------------

void GCApriorGPU::Send( const Freesurfer::GCAlinearPrior& src )
{

  // Allocate memory
  this->Allocate( src.xDim, src.yDim, src.zDim, src.n4D );

  const size_t nVoxels = this->xDim * this->yDim * this->zDim;

  // Copy offsets array
  CUDA_SAFE_CALL( hipMemcpy( this->d_offsets4D,
                              &src.offsets4D.front(),
                              (nVoxels+1)*sizeof(size_t),
                              hipMemcpyHostToDevice ) );

  // Copy 3D arrays
  CUDA_SAFE_CALL( hipMemcpy( this->d_maxLabels,
                              &src.maxLabels.front(),
                              nVoxels*sizeof(short),
                              hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( this->d_totTraining,
                              &src.totTraining.front(),
                              nVoxels*sizeof(int),
                              hipMemcpyHostToDevice ) );

  // Copy 4D arrays
  CUDA_SAFE_CALL( hipMemcpy( this->d_labels,
                              &src.labels.front(),
                              this->n4D*sizeof(unsigned short),
                              hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( this->d_priors,
                              &src.priors.front(),
                              this->n4D*sizeof(float),
                              hipMemcpyHostToDevice ) );
}

}
}
