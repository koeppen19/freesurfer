
#include <hip/hip_runtime.h>
/**
 * @file  gcamregisterpipeline_cuda.cu
 * @brief Implementation of GCAMregisterPipeline for the GPU
 *
 * Reference:
  * "Whole Brain Segmentation: Automated Labeling of Neuroanatomical
  * Structures in the Human Brain", Fischl et al.
  * (2002) Neuron, 33:341-355.
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:07 $
 *    $Revision: 1.3.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */


#ifdef GCAMORPH_ON_GPU

#include "macros.h"
#include "error.h"

#include "gcamorph.h"

#include "chronometer.hpp"

#include "mriframegpu.hpp"
#include "gcamorphgpu.hpp"


#include "gcamorphenergy.hpp"
#include "gcamregisterlevel_cuda.hpp"


// ========================================================================

template<typename T, typename U>
void RegisterPipeline( GPU::Classes::GCAmorphGPU& gcam,
                       const GPU::Classes::MRIframeGPU<T>& mri,
                       const GPU::Classes::MRIframeGPU<U>& mri_smooth,
                       GCA_MORPH_PARMS *parms,
                       double *last_rms,
                       int *level_steps,
                       int i )
{
  GPU::Algorithms::GCAmorphEnergy gcamEnergy;

  *last_rms = gcamEnergy.ComputeRMS( gcam, mri, parms );
  if( i==0 )
  {
    parms->start_rms = *last_rms;
  }
  *level_steps = parms->start_t;
  RegisterLevel( gcam, mri, mri_smooth, parms);
}


// ================

template<typename T, typename U>
void
gcamRPfinalDispatch( GCA_MORPH *gcam,
                     MRI *mri,
                     MRI *mri_smooth,
                     GCA_MORPH_PARMS *parms,
                     double *last_rms,
                     int *level_steps,
                     int i )
{
  GPU::Classes::GCAmorphGPU myGCAM;
  GPU::Classes::MRIframeGPU<T> myMRI;
  GPU::Classes::MRIframeGPU<U> myMRIsmooth;

  // Handle the MRIs
  myMRI.Allocate( mri );
  myMRI.Send( mri, 0 );

  myMRIsmooth.Allocate( mri_smooth );
  myMRIsmooth.Send( mri_smooth, 0 );

  // Put the GCAM on the GPU
  myGCAM.CheckIntegrity(); // Shouldn't be necessary....
  myGCAM.SendAll( gcam );

  // Run the computation
  RegisterPipeline( myGCAM, myMRI, myMRIsmooth, parms,
                    last_rms, level_steps, i);

  // Retrieve results
  myGCAM.RecvAll( gcam );
}


// -----------


template<typename T>
void
gcamRPsmoothDispatch(  GCA_MORPH *gcam,
                       MRI *mri,
                       MRI *mri_smooth,
                       GCA_MORPH_PARMS *parms,
                       double *last_rms,
                       int *level_steps,
                       int i )
{
  switch( mri_smooth->type )
  {

  case MRI_UCHAR:
    gcamRPfinalDispatch<T,unsigned char>( gcam, mri, mri_smooth, parms,
                                          last_rms, level_steps, i );
    break;
  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised type for mri_smooth "
              << mri_smooth->type << std::endl;
    abort();
  }

}


// -------------------

void GCAMregisterPipelineGPU( GCA_MORPH *gcam,
                              MRI *mri,
                              MRI *mri_smooth,
                              GCA_MORPH_PARMS *parms,
                              double *last_rms,
                              int *level_steps,
                              int i )
{

  switch( mri->type )
  {

  case MRI_UCHAR:
    gcamRPsmoothDispatch<unsigned char>( gcam, mri, mri_smooth, parms,
                                         last_rms, level_steps, i );
    break;

  case MRI_FLOAT:
    gcamRPsmoothDispatch<float>( gcam, mri, mri_smooth, parms,
                                 last_rms, level_steps, i );
    break;

  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised type for mri "
              << mri->type << std::endl;
    abort();
  }

}


#endif
