/**
 * @file  affinegpu.cu
 * @brief Holds affine transformation class for the GPU
 *
 * Holds an affine transformation type for the GPU
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:07 $
 *    $Revision: 1.5.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include <cstdlib>


#include <iostream>



#include "affinegpu.hpp"

// ====================================================

namespace GPU
{
namespace Classes
{

AffineTransformation::AffineTransformation( void )
{
  /*!
  Default constructor zeros the matrix
  */

  // Have a little sanity check
  if( kVectorSize != 4 )
  {
    std::cerr << __FUNCTION__
              << ": Incompatible universe detected"
              << std::endl
              << "Please adjust number of spatial dimensions "
              << "to 3 and try again"
              << std::endl;
    exit( EXIT_FAILURE );
  }

  for( unsigned int i=0; i<kMatrixSize; i++ )
  {
    this->matrix[i] = 0;
  }
}

AffineTransformation::AffineTransformation( const MATRIX* src )
{
  /*!
  Constructor to take values from a real 4x4 matrix.
  Double checks everything before performing assignment
  */

  if( src->type != MATRIX_REAL )
  {
    std::cerr << __FUNCTION__ << ": Invalid matrix type " <<
              src->type << std::endl;
    exit( EXIT_FAILURE );
  }

  if( static_cast<unsigned int>(src->rows) != kVectorSize )
  {
    std::cerr << __FUNCTION__ << ": Invalid number of rows " <<
              src->rows << std:: endl;
    exit( EXIT_FAILURE );
  }

  if( static_cast<unsigned int>(src->cols) != kVectorSize )
  {
    std::cerr << __FUNCTION__ << ": Invalid number of cols " <<
              src->cols << std:: endl;
    exit( EXIT_FAILURE );
  }

  // Do the copy
  for( unsigned int i=0; i<kVectorSize; i++ )
  {
    for( unsigned int j=0; j<kVectorSize; j++ )
    {
      this->operator()( i, j ) = *MATRIX_RELT( src, i+1, j+1 );
    }
  }
}



void AffineTransformation::SetTransform( const MATRIX* src )
{
  /*!
  Duplicates the functionality of the constructor from
  a real 4x4 matrix
  */
  if( src->type != MATRIX_REAL )
  {
    std::cerr << __FUNCTION__ << ": Invalid matrix type " <<
              src->type << std::endl;
    exit( EXIT_FAILURE );
  }

  if( static_cast<unsigned int>(src->rows) != kVectorSize )
  {
    std::cerr << __FUNCTION__ << ": Invalid number of rows " <<
              src->rows << std:: endl;
    exit( EXIT_FAILURE );
  }

  if( static_cast<unsigned int>(src->cols) != kVectorSize )
  {
    std::cerr << __FUNCTION__ << ": Invalid number of cols " <<
              src->cols << std:: endl;
    exit( EXIT_FAILURE );
  }

  // Do the copy
  for( unsigned int i=0; i<kVectorSize; i++ )
  {
    for( unsigned int j=0; j<kVectorSize; j++ )
    {
      this->operator()( i, j ) = *MATRIX_RELT( src, i+1, j+1 );
    }
  }
}

void AffineTransformation::SetTransform( const AffineMatrix* src )
{
  memcpy( this->matrix, src->mat, kMatrixSize*sizeof(float) );
}


}
}
