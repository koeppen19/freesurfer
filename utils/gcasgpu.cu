/**
 * @file  gcasgpu.cu
 * @brief Holds GCAS class for the GPU
 *
 *
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:07 $
 *    $Revision: 1.6.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */

using namespace std;

#include "cudacheck.h"

#include "gcasgpu.hpp"



// ====================================================

namespace GPU
{

namespace Classes
{


void GCASampleGPU::SendGPU( GCA *gca,
                            const GCA_SAMPLE *gcaSample,
                            const MRI *mri_inputs,
                            const int nSamples )
{
  /*!
  Sends the given GCAS to the GPU, doing
  various other things whose use is currently
  obscure.
  */

  // Verify the number of inputs
  if( gca->ninputs != 1 )
  {
    cerr << __FUNCTION__
         << ": Can only have one input"
         << endl;
    exit( EXIT_FAILURE );
  }

  // Allocate device memory
  this->Allocate( nSamples );

  // Identity transform matrix
  MATRIX *identity = MatrixIdentity( 4, NULL );
  TRANSFORM *identityTransform = TransformAlloc( LINEAR_VOX_TO_VOX, NULL );
  static_cast<LTA*>(identityTransform->xform)->xforms[0].m_L = identity;

  // Allocate some arrays
  int* myx = new int[nSamples];
  int* myy = new int[nSamples];
  int* myz = new int[nSamples];

  float* covars = new float[nSamples];
  float* priors = new float[nSamples];
  float* means = new float[nSamples];

  for( int i=0; i<nSamples; i++ )
  {
    // Copy code from GCAcomputeLogSampleProbability() in gca.c
    int xp = gcaSample[i].xp;
    int yp = gcaSample[i].yp;
    int zp = gcaSample[i].zp;

    if( GCApriorToSourceVoxel( gca, mri_inputs, identityTransform,
                               xp, yp, zp,
                               &myx[i], &myy[i], &myz[i] ) != NO_ERROR )
    {
      cerr << __FUNCTION__ << ": Failed with i=" << i << endl;
      exit( EXIT_FAILURE );
    }

    // These lines require the check for ninputs==1
    covars[i] = gcaSample[i].covars[0];
    means[i] = gcaSample[i].means[0];
    priors[i] = gcaSample[i].prior;

  }

  // Send to the GPU
  CUDA_SAFE_CALL( hipMemcpy( this->d_x, myx,
                              nSamples*sizeof(int),
                              hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( this->d_y, myy,
                              nSamples*sizeof(int),
                              hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( this->d_z, myz,
                              nSamples*sizeof(int),
                              hipMemcpyHostToDevice ) );

  CUDA_SAFE_CALL( hipMemcpy( this->d_means, means,
                              nSamples*sizeof(float),
                              hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( this->d_covars, covars,
                              nSamples*sizeof(float),
                              hipMemcpyHostToDevice ) );
  CUDA_SAFE_CALL( hipMemcpy( this->d_priors, priors,
                              nSamples*sizeof(float),
                              hipMemcpyHostToDevice ) );

  // Release memory
  delete[] myx;
  delete[] myy;
  delete[] myz;
  delete[] covars;
  delete[] priors;
  delete[] means;

  // Following should also free the identity matrix
  TransformFree( &identityTransform );
}

// ##################################################

// Memory management

void GCASampleGPU::Allocate( const unsigned int n )
{
  if( this->nSamplesAlloc < n )
  {
    this->Release();

    CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_x),
                                n*sizeof(int) ) );
    CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_y),
                                n*sizeof(int) ) );
    CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_z),
                                n*sizeof(int) ) );

    CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_means),
                                n*sizeof(float) ) );
    CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_priors),
                                n*sizeof(float) ) );
    CUDA_SAFE_CALL( hipMalloc( (void**)&(this->d_covars),
                                n*sizeof(float) ) );

    this->nSamplesAlloc = n;
  }

  this->nSamples = n;
}

void GCASampleGPU::Release( void )
{
  if( this->nSamplesAlloc != 0 )
  {
    hipFree( d_x );
    hipFree( d_y );
    hipFree( d_z );
    hipFree( d_means );
    hipFree( d_priors );
    hipFree( d_covars );
    this->nSamples = 0;
    this->nSamplesAlloc = 0;
  }
}

}
}
