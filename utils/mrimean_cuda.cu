#include "hip/hip_runtime.h"
/**
 * @file  mrimean_cuda.cu
 * @brief Holds MRI mean function for the GPU
 *
 * Implements MRI mean function on the GPU
 */
/*
 * Original Author: Richard Edgar
 * CVS Revision Info:
 *    $Author: nicks $
 *    $Date: 2012/12/14 10:35:08 $
 *    $Revision: 1.21.2.1 $
 *
 * Copyright © 2011 The General Hospital Corporation (Boston, MA) "MGH"
 *
 * Terms and conditions for use, reproduction, distribution and contribution
 * are found in the 'FreeSurfer Software License Agreement' contained
 * in the file 'LICENSE' found in the FreeSurfer distribution, and here:
 *
 * https://surfer.nmr.mgh.harvard.edu/fswiki/FreeSurferSoftwareLicense
 *
 * Reporting: freesurfer@nmr.mgh.harvard.edu
 *
 */

#include <cstdlib>
#include <cstdio>

#include <iostream>
#include <iomanip>


#include "chronometer.hpp"
#include "cudacheck.h"


#include "mriframegpu.hpp"

#include "mrimean_cuda.hpp"

#include "mrimean_cuda.h"






namespace GPU
{
namespace Algorithms
{

const unsigned int kMRImeanBlockSize = 16;

//! Min function for ints
__device__ int min( const int& a, const int& b )
{
  if( a < b )
  {
    return( a );
  }
  else
  {
    return( b );
  }
}

//! Max function for ints
__device__ int max( const int& a, const int& b )
{
  if( a > b )
  {
    return( a );
  }
  else
  {
    return( b );
  }
}


//! Kernel to compute x direction means
template<typename T>
__global__ void MRImeanX( const GPU::Classes::MRIframeOnGPU<T> src,
                          GPU::Classes::MRIframeOnGPU<float> dst,
                          const dim3 coverGrid,
                          const unsigned int wSize )
{
  /*!
  Kernel to compute means in the x direction, based on
  the given window size.
  Basically, does a 1D convolution, but with different
  boundary conditions to MRIConvolveKernelX.
  Also, since this is meant to be part of a pipeline,
  the destination type must be float
  */
  const unsigned int by = blockIdx.x / coverGrid.x;
  const unsigned int bx = blockIdx.x % coverGrid.x;
  const unsigned int tx = threadIdx.x;
  const unsigned int ty = threadIdx.y;

  const int ixStart = bx * kMRImeanBlockSize;
  const int iyStart = by * kMRImeanBlockSize;

  const int ix = ixStart + tx;
  const int iy = iyStart + ty;
  const int iz = blockIdx.y;

  const int wHalf = wSize/2;

  // Calculate voxels which will contribute, clamping to edges
  const unsigned int myxMin = max( 0           , ix - wHalf );
  const unsigned int myxMax = min( dst.dims.x-1, ix + wHalf );

  // Again, declare int to remove need for some casts
  const int patchSize = NextMultiple( max(wHalf,1), kMRImeanBlockSize );

  // Accumulator
  float myVal = 0;

  __shared__ float currPatch[kMRImeanBlockSize][kMRImeanBlockSize];

  // Calculate patch limits (note integer declarations avoid -ve trouble)
  const int xDimRound = NextMultiple( src.dims.x, kMRImeanBlockSize );
  const int xFirst = max( 0, ixStart - patchSize );
  const int xLast  = min( xDimRound - kMRImeanBlockSize,
                          ixStart + patchSize );

  for( int xBegin = xFirst;
       xBegin <= xLast;
       xBegin += kMRImeanBlockSize )
  {
    // Load the patch
    currPatch[ty][tx] = src( xBegin+tx, iy, iz );
    __syncthreads();

    // Accumulate desired values
    for( unsigned int i=0; i<kMRImeanBlockSize; i++ )
    {
      int actx = xBegin + i;

      if( (actx>=myxMin) && (actx<=myxMax) )
      {
        myVal += currPatch[ty][i];
      }

    }

    __syncthreads();
  }

  // Save result
  if( dst.InVolume( ix, iy, iz ) )
  {
    dst(ix,iy,iz) = dst.ConvertFloat( myVal );
  }
}


//! Kernel to compute y direction means
template<typename T>
__global__ void MRImeanY( const GPU::Classes::MRIframeOnGPU<T> src,
                          GPU::Classes::MRIframeOnGPU<float> dst,
                          const dim3 coverGrid,
                          const unsigned int wSize )
{
  /*!
  Kernel to compute means in the y direction, based on
  the given window size.
  Basically, does a 1D convolution, but with different
  boundary conditions to MRIConvolveKernelY.
  Also, since this is meant to be part of a pipeline,
  the destination type must be float
  */
  const unsigned int by = blockIdx.x / coverGrid.x;
  const unsigned int bx = blockIdx.x % coverGrid.x;
  const unsigned int tx = threadIdx.x;
  const unsigned int ty = threadIdx.y;

  const int ixStart = bx * kMRImeanBlockSize;
  const int iyStart = by * kMRImeanBlockSize;

  const int ix = ixStart + tx;
  const int iy = iyStart + ty;
  const int iz = blockIdx.y;

  const int wHalf = wSize/2;

  // Calculate voxels which will contribute, clamping to edges
  const unsigned int myyMin = max( 0           , iy - wHalf );
  const unsigned int myyMax = min( dst.dims.y-1, iy + wHalf );

  // Again, declare int to remove need for some casts
  const int patchSize = NextMultiple( max(wHalf,1), kMRImeanBlockSize );

  // Accumulator
  float myVal = 0;

  __shared__ float currPatch[kMRImeanBlockSize][kMRImeanBlockSize];

  // Calculate patch limits (note integer declarations avoid -ve trouble)
  const int yDimRound = NextMultiple( src.dims.y, kMRImeanBlockSize );

  const int yFirst = max( 0, iyStart - patchSize );
  const int yLast  = min( yDimRound - kMRImeanBlockSize,
                          iyStart + patchSize );

  for( int yBegin = yFirst;
       yBegin <= yLast;
       yBegin += kMRImeanBlockSize )
  {
    // Load the patch
    currPatch[ty][tx] = src( ix, yBegin+ty, iz );
    __syncthreads();

    // Accumulate desired values
    for( unsigned int i=0; i<kMRImeanBlockSize; i++ )
    {
      int acty = yBegin + i;

      if( (acty>=myyMin) && (acty<=myyMax) )
      {
        myVal += currPatch[i][tx];
      }

    }

    __syncthreads();
  }

  // Save result
  if( dst.InVolume( ix, iy, iz ) )
  {
    dst(ix,iy,iz) = dst.ConvertFloat( myVal );
  }
}


//! Kernel to compute z direction means
template<typename T>
__global__ void MRImeanZ( const GPU::Classes::MRIframeOnGPU<T> src,
                          GPU::Classes::MRIframeOnGPU<float> dst,
                          const dim3 coverGrid,
                          const unsigned int wSize )
{
  /*!
  Kernel to compute means in the x direction, based on
  the given window size.
  Basically, does a 1D convolution, but with different
  boundary conditions to MRIConvolveKernelZ.
  Also, since this is meant to be part of a pipeline,
  the destination type must be float
  */
  const unsigned int bz = blockIdx.x / coverGrid.x;
  const unsigned int bx = blockIdx.x % coverGrid.x;
  const unsigned int tx = threadIdx.x;
  // Note... tz is threadIdx.y
  const unsigned int tz = threadIdx.y;

  const int ixStart = bx * kMRImeanBlockSize;
  const int izStart = bz * kMRImeanBlockSize;

  const int ix = ixStart + tx;
  const int iy = blockIdx.y;
  const int iz = izStart + tz;

  const int wHalf = wSize/2;

  // Calculate voxels which will contribute, clamping to edges
  const unsigned int myzMin = max( 0           , iz - wHalf );
  const unsigned int myzMax = min( dst.dims.z-1, iz + wHalf );

  // Again, declare int to remove need for some casts
  const int patchSize = NextMultiple( max(wHalf,1), kMRImeanBlockSize );

  // Accumulator
  float myVal = 0;

  __shared__ float currPatch[kMRImeanBlockSize][kMRImeanBlockSize];

  // Calculate patch limits (note integer declarations avoid -ve trouble)
  const int zDimRound = NextMultiple( src.dims.z, kMRImeanBlockSize );

  const int zFirst = max( 0, izStart - patchSize );
  const int zLast  = min( zDimRound - kMRImeanBlockSize,
                          izStart + patchSize );

  for( int zBegin = zFirst;
       zBegin <= zLast;
       zBegin += kMRImeanBlockSize )
  {
    // Load the patch
    currPatch[tz][tx] = src( ix, iy, zBegin+tz );
    __syncthreads();

    // Accumulate desired values
    for( unsigned int i=0; i<kMRImeanBlockSize; i++ )
    {
      int actz = zBegin + i;

      if( (actz>=myzMin) && (actz<=myzMax) )
      {
        myVal += currPatch[i][tx];
      }

    }

    __syncthreads();
  }

  // Save result
  if( dst.InVolume( ix, iy, iz ) )
  {
    dst(ix,iy,iz) = dst.ConvertFloat( myVal );
  }
}


//! Kernel to normalise means
template<typename U>
__global__ void MRImeanNormal( const GPU::Classes::MRIframeOnGPU<float> src,
                               GPU::Classes::MRIframeOnGPU<U> dst,
                               const dim3 coverGrid,
                               const unsigned int wSize )
{
  /*!
  Kernel to normalise the means computed by the earlier
  stages.
  As such, the input type must be a float
  */
  const unsigned int by = blockIdx.x / coverGrid.x;
  const unsigned int bx = blockIdx.x % coverGrid.x;
  const unsigned int tx = threadIdx.x;
  const unsigned int ty = threadIdx.y;

  const int ixStart = bx * kMRImeanBlockSize;
  const int iyStart = by * kMRImeanBlockSize;

  const int ix = ixStart + tx;
  const int iy = iyStart + ty;
  const int iz = blockIdx.y;

  const int wHalf = wSize/2;

  // Calculate voxels which contributed, clamping to edges
  const unsigned int myxMin = max( 0           , ix - wHalf );
  const unsigned int myxMax = min( dst.dims.x-1, ix + wHalf );
  const unsigned int myyMin = max( 0           , iy - wHalf );
  const unsigned int myyMax = min( dst.dims.y-1, iy + wHalf );
  const unsigned int myzMin = max( 0           , iz - wHalf );
  const unsigned int myzMax = min( dst.dims.z-1, iz + wHalf );


  const unsigned long myVolume = ( myxMax - myxMin + 1 ) *
                                 (myyMax - myyMin + 1 ) *
                                 (myzMax - myzMin + 1 );

  if( dst.InVolume( ix, iy, iz ) )
  {
    dst( ix, iy, iz ) = dst.ConvertFloat( src( ix, iy, iz ) / myVolume );
  }
}






// ######################################################

// Define the static timers
SciGPU::Utilities::Chronometer MRImean::tMem, MRImean::tHostMem;
SciGPU::Utilities::Chronometer MRImean::tSend, MRImean::tRecv;
SciGPU::Utilities::Chronometer MRImean::tCompute;
SciGPU::Utilities::Chronometer MRImean::tTotal;

// =======================

template<typename T>
void MRImean::DispatchWrap( const MRI* src, MRI* dst,
                            const unsigned int wSize,
                            const int srcFrame,
                            const int dstFrame ) const
{
  switch( dst->type )
  {
  case MRI_UCHAR:
    this->MeanDispatch<T,unsigned char>( src, dst, wSize,
                                         srcFrame, dstFrame );
    break;

  case MRI_SHORT:
    this->MeanDispatch<T,short>( src, dst, wSize, srcFrame, dstFrame );
    break;

  case MRI_FLOAT:
    this->MeanDispatch<T,float>( src, dst, wSize, srcFrame, dstFrame );
    break;

  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised destination MRI type "
              << dst->type
              << std::endl;
    exit( EXIT_FAILURE );
  }
}

// =========================


void MRImean::Allocate( const size_t nBytes ) const
{
  if( this->workSize < nBytes )
  {
    this->Release();

    MRImean::tHostMem.Start();
    CUDA_SAFE_CALL( hipHostAlloc( (void**)&(this->h_workspace),
                                   nBytes,
                                   hipHostMallocDefault ) );
    this->workSize = nBytes;
    MRImean::tHostMem.Stop();
  }
}


//! Releases internal pinned memory buffer
void MRImean::Release( void ) const
{
  if( h_workspace != NULL )
  {
    MRImean::tHostMem.Start();
    hipHostFree( h_workspace );
    h_workspace = NULL;
    workSize = 0;
    MRImean::tHostMem.Stop();
  }
}


// ========================================================



MRImean::~MRImean( void )
{
  this->Release();
}


void MRImean::ShowTimings( void )
{
#ifdef CUDA_SHOW_TIMINGS
  std::cout << "==================================" << std::endl;
  std::cout << "GPU Mean timers" << std::endl;
  std::cout << "---------------" << std::endl;
#ifndef CUDA_FORCE_SYNC
  std::cout << "WARNING: CUDA_FORCE_SYNC not #defined" << std::endl;
  std::cout << "Timings may not be accurate" << std::endl;
#endif
  std::cout << std::endl;

  std::cout << "Host Memory : " << MRImean::tHostMem << std::endl;
  std::cout << "GPU Memory  : " << MRImean::tMem << std::endl;
  std::cout << "Send        : " << MRImean::tSend << std::endl;
  std::cout << "Compute     : " << MRImean::tCompute << std::endl;
  std::cout << "Receive     : " << MRImean::tRecv << std::endl;
  std::cout << "--------------" << std::endl;
  std::cout << "Total : " << MRImean::tTotal << std::endl;
  std::cout << "==================================" << std::endl;
#endif
}




void MRImean::DoMean( const MRI* src, MRI* dst,
                      const unsigned int wSize,
                      const unsigned int srcFrame,
                      const unsigned int dstFrame ) const
{
  switch( src->type )
  {
  case MRI_UCHAR:
    this->DispatchWrap<unsigned char>( src, dst, wSize,
                                       srcFrame, dstFrame );
    break;

  case MRI_SHORT:
    this->DispatchWrap<short>( src, dst, wSize, srcFrame, dstFrame );
    break;

  case MRI_FLOAT:
    this->DispatchWrap<float>( src, dst, wSize, srcFrame, dstFrame );
    break;

  default:
    std::cerr << __FUNCTION__
              << ": Unrecognised source MRI type "
              << src->type
              << std::endl;
    exit( EXIT_FAILURE );
  }
}


//! Templated dispatch for known data types
template<typename T, typename U>
void MRImean::MeanDispatch( const MRI* src, MRI* dst,
                            const unsigned int wSize,
                            const int srcFrame,
                            const int dstFrame ) const
{
  /*!
  Templated dispatch routine for MRI mean function on the
  GPU.
  Transfers data to the GPU, and retrieves the results
  */
  MRImean::tTotal.Start();

  GPU::Classes::MRIframeGPU<T> srcGPU;
  GPU::Classes::MRIframeGPU<U> dstGPU;

  size_t srcWorkSize, dstWorkSize;

  // Allocate the GPU arrays
  MRImean::tMem.Start();
  srcGPU.Allocate( src );
  dstGPU.Allocate( dst );
  MRImean::tMem.Stop();

  // Put in some sanity checks
  srcGPU.VerifyMRI( src );
  dstGPU.VerifyMRI( dst );

  // Allocate the workspace array
  srcWorkSize = srcGPU.BufferSize();
  dstWorkSize = dstGPU.BufferSize();

  if( srcWorkSize > dstWorkSize )
  {
    this->Allocate( srcWorkSize );
  }
  else
  {
    this->Allocate( dstWorkSize );
  }

  // Send the source data
  MRImean::tSend.Start();
  srcGPU.SendFrame( src, srcFrame, this->h_workspace, this->stream );
  MRImean::tSend.Stop();

  // Run the filter
  this->RunGPU( srcGPU, dstGPU, wSize );

  // Get the results
  MRImean::tRecv.Start();
  dstGPU.RecvFrame( dst, dstFrame, this->h_workspace, this->stream );
  MRImean::tRecv.Stop();

  CUDA_CHECK_ERROR( "Mean filtering failure" );

  MRImean::tTotal.Stop();
}



//! Runs the mean filtering kernel
template<typename T, typename U>
void MRImean::RunGPU( const GPU::Classes::MRIframeGPU<T> &src,
                      GPU::Classes::MRIframeGPU<U> &dst,
                      const unsigned int wSize ) const
{
  /*!
  Runs the mean filtering kernel on the GPU.
  Assumes most things are properly set already
  */
  const dim3 srcDims = src.GetDims();
  const dim3 dstDims = dst.GetDims();

  // Check dimensions
  if( srcDims != dstDims )
  {
    std::cerr << __FUNCTION__ << ": Dimension mismatch"
              << std::endl;
    exit( EXIT_FAILURE );
  }


  // We need intermediates which are floats
  GPU::Classes::MRIframeGPU<float> f1, f2;

  // Get correctly sized intermediates
  MRImean::tMem.Start();
  f1.Allocate( src );
  f2.Allocate( src );
  MRImean::tMem.Stop();


  // Do the three convolutions. Recall objects have same dims
  dim3 grid, threads;


  threads.x = threads.y = kMRImeanBlockSize;
  threads.z = 1;

  const dim3 coverGrid = dst.CoverBlocks( kMRImeanBlockSize );

  grid.x = coverGrid.x * coverGrid.y;
  grid.y = dstDims.z;
  grid.z = 1;

  MRImean::tCompute.Start();

  // Do the X direction
  MRImeanX<T>
  <<<grid,threads,0,this->stream>>>
  ( src, f1, coverGrid, wSize );
  CUDA_CHECK_ERROR_ASYNC( "MRImeanX kernel failed" );

  // Do the Y direction
  MRImeanY<float>
  <<<grid,threads,0,this->stream>>>
  ( f1, f2, coverGrid, wSize );
  CUDA_CHECK_ERROR_ASYNC( "MRImeanY kernel failed" );

  // Slight change for Z direction
  grid.x = coverGrid.x * coverGrid.z;
  grid.y = dstDims.y;
  MRImeanZ<float>
  <<<grid,threads,0,this->stream>>>
  ( f2, f1, coverGrid, wSize );
  CUDA_CHECK_ERROR_ASYNC( "MRImeanZ kernel failed" );

  // Normalise
  grid.x = coverGrid.x * coverGrid.y;
  grid.y = dstDims.z;
  MRImeanNormal<U>
  <<<grid,threads,0,this->stream>>>
  ( f1, dst, coverGrid, wSize );
  CUDA_CHECK_ERROR_ASYNC( "MRImeanNormal failed!" );

  MRImean::tCompute.Stop();
}

}
}


static GPU::Algorithms::MRImean myMean;


MRI* MRImean_cuda( const MRI* src, MRI* dst,
                   const unsigned int wSize )
{
  /*!
    Wrapper around the GPU routine, to be called from the
    original MRImean routine.
    Note that the frames default to zero, per the original
    MRImean routine.
  */

  myMean.DoMean( src, dst, wSize );

  return( dst );
}
